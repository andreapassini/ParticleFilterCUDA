#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

// Not recommended by NVIDIA for unpredictable side effects but it works
//for __syncthreads() and CHECK
#ifndef __HIPCC__ 
#define __HIPCC__
#endif
#include <hip/device_functions.h>

#include "common.h"
#include "Particle.h"

#include <stdio.h>
#include <cmath>
#include <string>

#define PI 3.141592f

#define IDX(i,j,n) (i*n+j)
#define ABS(x,y) (x-y>=0?x-y:y-x)

#define DIM 100

#define BLOCKSIZE 1024  // block dim 1D
#define NUMBLOCKS 1024  // grid dim 1D 
#define N (NUMBLOCKS * BLOCKSIZE)


hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);



__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

__global__ void GenerateParticles(Particles* D_in, Particles* C_out, hiprandState* states, float2 x_range, float2 y_range, float2 heading_range) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    hiprand_init(tid, 0, 0, &states[tid]);
    float pos_x = x_range.x + ((x_range.y - x_range.x) * hiprand_uniform(&states[tid]));
    float pos_y = y_range.x + ((y_range.y - y_range.x) * hiprand_uniform(&states[tid]));
    float heading = hiprand_uniform(&states[tid]);
    heading = std::fmod(heading, 2 * PI);

    C_out->x[tid] = pos_x;
    C_out->y[tid] = pos_y;
    C_out->heading[tid] = heading;
}

//# def predict(particles, u, std, dt=1.):
//# """ move according to control input u (heading change, velocity)
//# with noise Q (std heading change, std velocity)`"""
__global__ void Predict(Particles* D_in, Particles* C_out, hiprandState* states, float* u, float* std, float dt) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    dt = 1.0f;

    hiprand_init(tid, 0, 0, &states[tid]);

    //# update heading
    float heading = D_in->heading[tid];
    heading += u[0] + hiprand_normal(&states[tid]);
    heading = std::fmod(heading, 2 * PI);
        
    //# move in the (noisy) commanded direction
    float dist = (u[1] * dt) + (hiprand_uniform(&states[tid]) * std[1]);
    float pos_x = D_in->x[tid];
    float pos_y = D_in->y[tid];
    pos_x += std::cos(heading) * dist;
    pos_y += std::sin(heading) * dist;

    C_out->x[tid] = pos_x;
    C_out->y[tid] = pos_y;
    C_out->heading[tid] = heading;
}

/*
 *  Device function: block parallel reduction based on warp unrolling
 */
__device__ void blockWarpUnroll(float* thisBlock, int blockDim, uint tid) {
    // in-place reduction in global memory
    for (int stride = blockDim / 2; stride > 32; stride >>= 1) {
        if (tid < stride)
            thisBlock[tid] += thisBlock[tid + stride];

        // synchronize within threadblock
        __syncthreads();
    }

    // unrolling warp
    if (tid < 32) {
        volatile float* vmem = thisBlock;
        vmem[tid] += vmem[tid + 32];
        vmem[tid] += vmem[tid + 16];
        vmem[tid] += vmem[tid + 8];
        vmem[tid] += vmem[tid + 4];
        vmem[tid] += vmem[tid + 2];
        vmem[tid] += vmem[tid + 1];
    }
}

__global__ void Norm_BlockUnroll8(float* in, float* out, float add, ulong n) {
    uint tid = threadIdx.x;
    ulong idx = blockIdx.x * blockDim.x + threadIdx.x;

    // boundary check
    if (idx >= n)
        return;

    // convert global data pointer to the local pointer of this block
    float* thisBlock = in + blockIdx.x * blockDim.x * 8;

    float a = 0.0f;
    float temp = 0.0f;

    // unrolling 8 blocks
    if (idx + 7 * blockDim.x < n) {
        for (int i = 0; i < 8; i++) {
            temp = in[idx + i * blockDim.x];
            temp += add;
            a += temp * temp;
        }
        in[idx] = a;
    }

    __syncthreads();

    // block parall. reduction based on warp unrolling 
    blockWarpUnroll(thisBlock, blockDim.x, tid);

    // write result for this block to global mem
    if (tid == 0)
        out[blockIdx.x] = thisBlock[0];
}

//# def update(particles, weights, z, R, landmarks):
__global__ void Update(float2 norm, Particles* particles, Particles* C_out, float* weights, float* z, float* landmarks, int numberOfLandmarks, float R) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // boundary check
    if (tid >= DIM)
        return;

    //    weights *= scipy.stats.norm(distance, R).pdf(z[i]) // norm.pdf(x) = exp(-x**2/2)/sqrt(2*pi)


    //weights += 1.e-300      # avoid round - off to zero
    //weights /= sum(weights) # normalize

}

void particleFilter(Particles* p);

int main()
{
    Particles p;

    CreateParticleDim(&p, DIM);

    particleFilter(&p);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    CHECK(hipDeviceReset());

    return 0;
}

void particleFilter(Particles* p) {

    Particles d_p;

    CHECK(hipMalloc((void**)&d_p.x, DIM * sizeof(float)));
    CHECK(hipMemcpy(d_p.x, p->x, DIM * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMalloc((void**)&d_p.y, DIM * sizeof(float)));
    CHECK(hipMemcpy(d_p.y, p->y, DIM * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMalloc((void**)&d_p.weights, DIM * sizeof(float)));
    CHECK(hipMemcpy(d_p.weights, p->weights, DIM * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMalloc((void**)&d_p.heading, DIM * sizeof(float)));
    CHECK(hipMemcpy(d_p.heading, p->heading, DIM * sizeof(float), hipMemcpyHostToDevice));

    // OUTPUT for Device
    Particles d_out;
    CHECK(hipMalloc((void**)&d_out.x, DIM * sizeof(float)));
    CHECK(hipMemcpy(d_out.x, p->x, DIM * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMalloc((void**)&d_out.y, DIM * sizeof(float)));
    CHECK(hipMemcpy(d_out.y, p->y, DIM * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMalloc((void**)&d_out.weights, DIM * sizeof(float)));
    CHECK(hipMemcpy(d_out.weights, p->weights, DIM * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMalloc((void**)&d_out.heading, DIM * sizeof(float)));
    CHECK(hipMemcpy(d_out.heading, p->heading, DIM * sizeof(float), hipMemcpyHostToDevice));



    hipFree(d_p.x);
    hipFree(d_p.y);
    hipFree(d_p.heading);
    hipFree(d_p.weights);
    hipFree(d_out.x);
    hipFree(d_out.y);
    hipFree(d_out.heading);
    hipFree(d_out.weights);
}

void euclideanNorm(Particles* p, float2* norm, float2* landmark) {

    Particles d_p;

    CHECK(hipMalloc((void**)&d_p.x, DIM * sizeof(float)));
    CHECK(hipMemcpy(d_p.x, p->x, DIM * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMalloc((void**)&d_p.y, DIM * sizeof(float)));
    CHECK(hipMemcpy(d_p.y, p->y, DIM * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMalloc((void**)&d_p.weights, DIM * sizeof(float)));
    CHECK(hipMemcpy(d_p.weights, p->weights, DIM * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMalloc((void**)&d_p.heading, DIM * sizeof(float)));
    CHECK(hipMemcpy(d_p.heading, p->heading, DIM * sizeof(float), hipMemcpyHostToDevice));


    // OUTPUT for Device
    Particles d_out;
    CHECK(hipMalloc((void**)&d_out.x, DIM * sizeof(float)));
    CHECK(hipMemcpy(d_out.x, p->x, DIM * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMalloc((void**)&d_out.y, DIM * sizeof(float)));
    CHECK(hipMemcpy(d_out.y, p->y, DIM * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMalloc((void**)&d_out.weights, DIM * sizeof(float)));
    CHECK(hipMemcpy(d_out.weights, p->weights, DIM * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMalloc((void**)&d_out.heading, DIM * sizeof(float)));
    CHECK(hipMemcpy(d_out.heading, p->heading, DIM * sizeof(float), hipMemcpyHostToDevice));


    Particles p_norm;

    CreateParticleDim(&p_norm, DIM);

    Norm_BlockUnroll8<<<NUMBLOCKS / 8, BLOCKSIZE >>>(d_p.x, d_out.x, -landmark->x, N);  // ERRROR at <<< can be simply ignored
    Norm_BlockUnroll8<<<NUMBLOCKS / 8, BLOCKSIZE >>>(d_p.y, d_out.y, -landmark->y, N);

    CHECK(hipDeviceSynchronize());
    CHECK(hipGetLastError());
    
    CHECK(hipMemcpy(p_norm.x, d_out.x, DIM * sizeof(float), hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(p_norm.y, d_out.y, DIM * sizeof(float), hipMemcpyDeviceToHost));

    // Sum on CPU the last elements
    for (uint i = 0; i < NUMBLOCKS / 8; i++) {
        norm->x += p_norm.x[i];
        norm->y += p_norm.y[i];
    }

    hipFree(d_p.x);
    hipFree(d_p.y);
    hipFree(d_p.heading);
    hipFree(d_p.weights);
    hipFree(d_out.x);
    hipFree(d_out.y);
    hipFree(d_out.heading);
    hipFree(d_out.weights);
    free(p_norm.x);
    free(p_norm.y);
    free(p_norm.heading);
    free(p_norm.weights);
}

//// Helper function for using CUDA to add vectors in parallel.
//hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
//{
//    int *dev_a = 0;
//    int *dev_b = 0;
//    int *dev_c = 0;
//    hipError_t cudaStatus;
//
//    //// Choose which GPU to run on, change this on a multi-GPU system.
//    cudaStatus = hipSetDevice(0);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
//        goto Error;
//    }
//
//    // Allocate GPU buffers for three vectors (two input, one output)    .
//    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMalloc failed!");
//        goto Error;
//    }
//
//    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMalloc failed!");
//        goto Error;
//    }
//
//    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMalloc failed!");
//        goto Error;
//    }
//
//    // Copy input vectors from host memory to GPU buffers.
//    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMemcpy failed!");
//        goto Error;
//    }
//
//    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMemcpy failed!");
//        goto Error;
//    }
//
//    // Launch a kernel on the GPU with one thread for each element.
//    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);
//
//    // Check for any errors launching the kernel
//    cudaStatus = hipGetLastError();
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
//        goto Error;
//    }
//    
//    // hipDeviceSynchronize waits for the kernel to finish, and returns
//    // any errors encountered during the launch.
//    cudaStatus = hipDeviceSynchronize();
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
//        goto Error;
//    }
//
//    // Copy output vector from GPU buffer to host memory.
//    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMemcpy failed!");
//        goto Error;
//    }
//
//Error:
//    hipFree(dev_c);
//    hipFree(dev_a);
//    hipFree(dev_b);
//    
//    return cudaStatus;
//}

