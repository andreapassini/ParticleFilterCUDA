#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

// Not recommended by NVIDIA for unpredictable side effects but it works
//for __syncthreads() and CHECK
#ifndef __HIPCC__ 
#define __HIPCC__
#endif

#include <hip/device_functions.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>

#include <stdio.h>
#include <cmath>
#include <string>
#include <time.h>

#include "common.h"
#include "Particle.h"
#include "Float2.h"

#include "MyDefs.h"

int THREADS_PER_BLOCK = BLOCKSIZE;
int ELEMENTS_PER_BLOCK = THREADS_PER_BLOCK * 2;

void scanLargeEvenDeviceArray(float* d_out, float* d_in, int length, bool bcao);
void scanLargeDeviceArray(float* d_out, float* d_in, int length, bool bcao);
void scanLargeEvenDeviceArray(float* d_out, float* d_in, int length, bool bcao);

/*
 *  Block by block parallel implementation without divergence (interleaved schema)
 */
__global__ void SumParReduce(float* in, float* out, ulong n) {

    uint tid = threadIdx.x;
    ulong idx = blockIdx.x * blockDim.x + threadIdx.x;

    // boundary check
    if (idx >= n)
        return;

    // convert global data pointer to the local pointer of this block
    float* thisBlock = in + blockIdx.x * blockDim.x;

    // in-place reduction in global memory
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride)
            thisBlock[tid] += thisBlock[tid + stride];

        // synchronize within threadblock
        __syncthreads();
    }

    // write result for this block to global mem
    if (tid == 0)
        out[blockIdx.x] = thisBlock[0];
}

/*
 *  Block by block parallel implementation without divergence (interleaved schema)
 */
__global__ void SumSquaredParReduce(float* in, float* out, const ulong n) {
    uint tid = threadIdx.x;
    ulong idx = blockIdx.x * blockDim.x + threadIdx.x;

    // boundary check
    if (idx >= n)
        return;

    // convert global data pointer to the local pointer of this block
    float* thisBlock = in + blockIdx.x * blockDim.x;

    // in-place reduction in global memory
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (stride == blockDim.x / 2) { // first time, multiply since we touch every single elements
            thisBlock[tid] = (thisBlock[tid] * thisBlock[tid]) * (thisBlock[tid + stride] * thisBlock[tid + stride]);
        }
        else if (tid < stride) {
            thisBlock[tid] += thisBlock[tid + stride];
        }

        // synchronize within threadblock
        __syncthreads();
    }

    // write result for this block to global mem
    if (tid == 0)
        out[blockIdx.x] = thisBlock[0];
}

__global__ void GenerateParticles(Particles* D_in, Particles* C_out, hiprandState* states, float2 x_range, float2 y_range, float2 heading_range) {
    uint tid = threadIdx.x;
    ulong idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= N)
        return;

    hiprand_init(idx, 0, 0, &states[idx]);
    float pos_x = x_range.x + ((x_range.y - x_range.x) * hiprand_uniform(&states[idx]));
    float pos_y = y_range.x + ((y_range.y - y_range.x) * hiprand_uniform(&states[idx]));
    float heading = heading_range.x + ((heading_range.y - heading_range.x) * hiprand_uniform(&states[idx]));
    heading = std::fmod(heading, 2 * PI);

    C_out->x[idx] = pos_x;
    C_out->y[idx] = pos_y;
    C_out->heading[idx] = heading;
}

static float SumArrayGPU(const float* const arrIn, const int dim, float* const sumOut) {
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float seconds = 0.0f;

    uint numBlocks = (dim + BLOCKSIZE - 1) / BLOCKSIZE;

    long blocksBytes = numBlocks * sizeof(float);
    long arrayBytes = dim * sizeof(float);

    float* arrOut, * d_arrIn, * d_arrOut;
    CHECK(hipMalloc((void**)&d_arrIn, arrayBytes));
    CHECK(hipMemcpy(d_arrIn, arrIn, arrayBytes, hipMemcpyHostToDevice));
    CHECK(hipMalloc((void**)&d_arrOut, blocksBytes));
    CHECK(hipMemset((void*)d_arrOut, 0, blocksBytes));
    arrOut = (float*)malloc(blocksBytes * sizeof(float));

    hipEventRecord(start);

    SumParReduce << <numBlocks, BLOCKSIZE >> > (d_arrIn, d_arrOut, dim);

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float iterMilliseconds = 0;
    hipEventElapsedTime(&iterMilliseconds, start, stop);
    float iterSeconds = iterMilliseconds / 1000.0f;
    seconds += iterSeconds;

    hipEventDestroy(start);
    hipEventDestroy(stop);

    //CHECK(hipDeviceSynchronize());
    CHECK(hipGetLastError());

    // memcopy D2H
    CHECK(hipMemcpy(arrOut, d_arrOut, blocksBytes, hipMemcpyDeviceToHost));

    // check result
    for (uint i = 0; i < numBlocks; i++) {
        *sumOut += arrOut[i];
    }

    hipFree(d_arrOut);
    hipFree(d_arrIn);
    free(arrOut);

    //CHECK(hipDeviceReset());

    return seconds;
}

__global__ void SumArrayWeightsGPUKernel(float* weights, float* posX, float* posY, float* outWeights, float* outPosY, float* outPosX, ulong n) {
    uint tid = threadIdx.x;
    ulong idx = blockIdx.x * blockDim.x + threadIdx.x;

    // boundary check
    if (idx >= n)
        return;

    // convert global data pointer to the local pointer of this block
    float* thisBlockWeights = weights + blockIdx.x * blockDim.x;
    float* thisBlockPosX = posX + blockIdx.x * blockDim.x;
    float* thisBlockPosY = posY + blockIdx.x * blockDim.x;

    // in-place reduction in global memory
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (stride == blockDim.x / 2) { // first time, multiply since we touch every single elements
            thisBlockPosX[tid] = (thisBlockPosX[tid] * thisBlockWeights[tid]) + (thisBlockPosX[tid + stride] * thisBlockWeights[tid + stride]);
            thisBlockPosY[tid] = (thisBlockPosY[tid] * thisBlockWeights[tid]) + (thisBlockPosY[tid + stride] * thisBlockWeights[tid + stride]);
            thisBlockWeights[tid] += thisBlockWeights[tid + stride];
        }
        else if (tid < stride) {
            thisBlockPosX[tid] += thisBlockPosX[tid + stride];
            thisBlockPosY[tid] += thisBlockPosY[tid + stride];
            thisBlockWeights[tid] += thisBlockWeights[tid + stride];
        }

        // synchronize within threadblock
        __syncthreads();
    }

    // write result for this block to global mem
    if (tid == 0) {
        outPosX[blockIdx.x] = thisBlockPosX[0];
        outPosY[blockIdx.x] = thisBlockPosY[0];
        outWeights[blockIdx.x] = thisBlockWeights[0];
    }

}
__global__ void SumArrayWeightsSqrdSubGPUKernel(float* weights, float* posX, float* posY, float* outWeights, float* outPosY,
    float* outPosX, ulong n, const float meanX, const float meanY) {
    uint tid = threadIdx.x;
    ulong idx = blockIdx.x * blockDim.x + threadIdx.x;

    // boundary check
    if (idx >= n)
        return;

    // convert global data pointer to the local pointer of this block
    float* thisBlockWeights = weights + blockIdx.x * blockDim.x;
    float* thisBlockPosX = posX + blockIdx.x * blockDim.x;
    float* thisBlockPosY = posY + blockIdx.x * blockDim.x;

    // in-place reduction in global memory
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (stride == blockDim.x / 2) { // first time, multiply since we touch every single elements
            thisBlockPosX[tid] = (thisBlockPosX[tid] - meanX) * (thisBlockPosX[tid] - meanX);
            thisBlockPosX[tid] = (thisBlockPosX[tid] * thisBlockWeights[tid]) + (thisBlockPosX[tid + stride] * thisBlockWeights[tid + stride]);

            thisBlockPosY[tid] = (thisBlockPosY[tid] - meanY) * (thisBlockPosY[tid] - meanY);
            thisBlockPosY[tid] = (thisBlockPosY[tid] * thisBlockWeights[tid]) + (thisBlockPosY[tid + stride] * thisBlockWeights[tid + stride]);

            thisBlockWeights[tid] += thisBlockWeights[tid + stride];
        }
        else if (tid < stride) {
            thisBlockPosX[tid] += thisBlockPosX[tid + stride];
            thisBlockPosY[tid] += thisBlockPosY[tid + stride];
            thisBlockWeights[tid] += thisBlockWeights[tid + stride];
        }

        // synchronize within threadblock
        __syncthreads();
    }

    // write result for this block to global mem
    if (tid == 0) {
        outPosX[blockIdx.x] = thisBlockPosX[0];
        outPosY[blockIdx.x] = thisBlockPosY[0];
        outWeights[blockIdx.x] = thisBlockWeights[0];
    }

}
float SumArrayWeightsGPU(const float* const posX, const float* const posY, float* weights,
    const int dim, float3* const sumsOut) {

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float seconds = 0.0f;

    float sumX = FLT_EPSILON;
    float sumY = FLT_EPSILON;
    float sumWeights = FLT_EPSILON;

    uint numBlocks = (dim + BLOCKSIZE - 1) / BLOCKSIZE;

    long blocksBytes = numBlocks * sizeof(float);
    long arrayBytes = dim * sizeof(float);

    float* d_weights, * d_weightsOut, * weightsOut;

    // To be removed
    float* a;
    CHECK(hipMalloc((void**)&a, sizeof(float)));

    float* d_posX, * d_posXOut, * posXOut;
    CHECK(hipMalloc((void**)&d_posX, arrayBytes));
    CHECK(hipMemcpy(d_posX, posX, arrayBytes, hipMemcpyHostToDevice));
    CHECK(hipMalloc((void**)&d_posXOut, blocksBytes));
    CHECK(hipMemset((void*)d_posXOut, 0, blocksBytes));
    posXOut = (float*)malloc(blocksBytes);

    CHECK(hipMalloc((void**)&d_weights, arrayBytes));
    CHECK(hipMemcpy(d_weights, weights, arrayBytes, hipMemcpyHostToDevice));
    CHECK(hipMalloc((void**)&d_weightsOut, blocksBytes));
    CHECK(hipMemset((void*)d_weightsOut, 0, blocksBytes));
    weightsOut = (float*)malloc(blocksBytes);

    float* d_posY, * d_posYOut, * posYOut;
    CHECK(hipMalloc((void**)&d_posY, arrayBytes));
    CHECK(hipMemcpy(d_posY, posY, arrayBytes, hipMemcpyHostToDevice));
    CHECK(hipMalloc((void**)&d_posYOut, blocksBytes));
    CHECK(hipMemset((void*)d_posYOut, 0, blocksBytes));
    posYOut = (float*)malloc(blocksBytes);

    hipEventRecord(start);

    SumArrayWeightsGPUKernel << <numBlocks, BLOCKSIZE >> > (d_weights, d_posX, d_posY, d_weightsOut, d_posXOut, d_posYOut, dim);

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    //CHECK(hipDeviceSynchronize());
    CHECK(hipGetLastError());

    float iterMilliseconds = 0.0f;
    hipEventElapsedTime(&iterMilliseconds, start, stop);
    float iterSeconds = iterMilliseconds / 1000.0f;
    seconds += iterSeconds;

    // memcopy D2H
    CHECK(hipMemcpy(weightsOut, d_weightsOut, blocksBytes, hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(posXOut, d_posXOut, blocksBytes, hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(posYOut, d_posYOut, blocksBytes, hipMemcpyDeviceToHost));

    // check result
    for (uint i = 0; i < numBlocks; i++) {
        sumX += posXOut[i];
        sumY += posYOut[i];
        sumWeights += weightsOut[i];
    }

    hipFree(d_weights);
    hipFree(d_weightsOut);
    free(weightsOut);

    hipFree(d_posX);
    hipFree(d_posXOut);
    free(posXOut);

    hipFree(d_posY);
    hipFree(d_posYOut);
    free(posYOut);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    //CHECK(hipDeviceReset());

    sumsOut->x = sumX;
    sumsOut->y = sumY;
    sumsOut->z = sumWeights;

    return seconds;
}
static float SumArrayWeightsSqrdSubGPU(const float* const posX, const float* const posY,
    const float* const weights, const int dim, const float meanX, const float meanY, float3* const sumsOut) {

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float seconds = 0.0f;

    float sumX = FLT_EPSILON;
    float sumY = FLT_EPSILON;
    float sumWeights = FLT_EPSILON;

    uint numBlocks = (dim + BLOCKSIZE - 1) / BLOCKSIZE;

    long blocksBytes = numBlocks * sizeof(float);
    long arrayBytes = dim * sizeof(float);

    float* d_weights, * d_weightsOut, * weightsOut;
    CHECK(hipMalloc((void**)&d_weights, arrayBytes));
    CHECK(hipMemcpy(d_weights, weights, arrayBytes, hipMemcpyHostToDevice));
    CHECK(hipMalloc((void**)&d_weightsOut, blocksBytes));
    CHECK(hipMemset((void*)d_weightsOut, 0, blocksBytes));
    weightsOut = (float*)malloc(blocksBytes * sizeof(float));

    float* d_posX, * d_posXOut, * posXOut;
    CHECK(hipMalloc((void**)&d_posX, arrayBytes));
    CHECK(hipMemcpy(d_posX, posX, arrayBytes, hipMemcpyHostToDevice));
    CHECK(hipMalloc((void**)&d_posXOut, blocksBytes));
    CHECK(hipMemset((void*)d_posXOut, 0, blocksBytes));
    posXOut = (float*)malloc(blocksBytes * sizeof(float));

    float* d_posY, * d_posYOut, * posYOut;
    CHECK(hipMalloc((void**)&d_posY, arrayBytes));
    CHECK(hipMemcpy(d_posY, posY, arrayBytes, hipMemcpyHostToDevice));
    CHECK(hipMalloc((void**)&d_posYOut, blocksBytes));
    CHECK(hipMemset((void*)d_posYOut, 0, blocksBytes));
    posYOut = (float*)malloc(blocksBytes * sizeof(float));

    hipEventRecord(start);

    SumArrayWeightsSqrdSubGPUKernel << <numBlocks, BLOCKSIZE >> > (d_weights, d_posX, d_posY, d_weightsOut, d_posXOut, d_posYOut, dim, meanX, meanY);

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float iterMilliseconds = 0;
    hipEventElapsedTime(&iterMilliseconds, start, stop);
    float iterSeconds = iterMilliseconds / 1000.0f;
    seconds += iterSeconds;

    hipEventDestroy(start);
    hipEventDestroy(stop);

    //CHECK(hipDeviceSynchronize());
    CHECK(hipGetLastError());

    // memcopy D2H
    CHECK(hipMemcpy(weightsOut, d_weightsOut, blocksBytes, hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(posXOut, d_posXOut, blocksBytes, hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(posYOut, d_posYOut, blocksBytes, hipMemcpyDeviceToHost));

    // check result
    for (uint i = 0; i < numBlocks; i++) {
        sumX += posXOut[i];
        sumY += posYOut[i];
        sumWeights += weightsOut[i];
    }

    hipFree(d_weights);
    hipFree(d_weightsOut);
    free(weightsOut);

    hipFree(d_posX);
    hipFree(d_posXOut);
    free(posXOut);

    hipFree(d_posY);
    hipFree(d_posYOut);
    free(posYOut);

    //CHECK(hipDeviceReset());

    sumsOut->x = sumX;
    sumsOut->y = sumY;
    sumsOut->z = sumWeights;

    return seconds;
}

// Norm Pdf
// https://docs.scipy.org/doc/scipy/reference/generated/scipy.stats.norm.html
// https://stackoverflow.com/questions/10847007/using-the-gaussian-probability-density-function-in-c
//  def normpdf(x, mu=0, sigma=1):
static float normpdf(const float x, const float mu = 0.0f, const float sigma = 1.0f) {
    //  u = float((x - mu) / abs(sigma))
    //  y = exp(-u * u / 2) / (sqrt(2 * pi) * abs(sigma))
    //  return y
    float u = (x - mu) / abs(sigma);
    float num = exp(-(u * u) / 2.0f);
    float y = num / (PI2SQRD * abs(sigma));
    return y;
}
__device__ float normPdfGPU(const float x, const float mu = 0.0f, const float sigma = 1.0f) {
    float u = (x - mu) / abs(sigma);
    float y = exp(-(u * u) / 2.0f) / (PI2SQRD * abs(sigma));
    return y;
}

float sqrdMagnitude(const float* const X, const int dim) {
    float sqrdMag = FLT_EPSILON;
    for (int i = 0; i < dim; i++) {
        sqrdMag += X[i] * X[i];
    }
    return sqrdMag;
}
static float magnitudeXY(const Particles* const p) {
    float sqrdMagX = sqrdMagnitude(p->x, N);
    float sqrdMagY = sqrdMagnitude(p->y, N);
    float magnitude = sqrt(sqrdMagX + sqrdMagY);
    return magnitude;
}
__device__ float MagnitudeGPU(const float x, const float y) {
    float mag = sqrt((x * x) + (y * y));
    return mag;
}
__device__ float MagnitudeGPU(const Float2* const vec2) {
    return MagnitudeGPU(vec2->x, vec2->y);

}
__device__ float MagnitudeGPU(const Float2 vec2) {
    return MagnitudeGPU(vec2.x, vec2.y);
}

static Float2 WeightedAverage(const Floats2* const pos, const float* const weights, const int dim) {
    Float2 avg;
    avg.x = 0.0f;
    avg.y = 0.0f;

    // numpy implementation: avg = sum(a * weights) / sum(weights)

    float sumWeights = FLT_EPSILON; // avoid div by 0
    Float2 sumPos;
    sumPos.x = 0.0f;
    sumPos.y = 0.0f;
    for (int i = 0; i < dim; i++) {
        sumWeights += weights[i];
        sumPos.x += pos->x[i] * weights[i];
        sumPos.y += pos->y[i] * weights[i];
    }

    avg.x = sumPos.x / sumWeights;
    avg.y = sumPos.y / sumWeights;

    return avg;
}
static float WeightedAverageGPU(const Floats2* const pos, float* weights, const int dim, Float2* const average) {
    float seconds = 0.0f;

    float3 sums;
    sums.x = 0.0f;
    sums.y = 0.0f;
    sums.z = 0.0f;

    seconds += SumArrayWeightsGPU(pos->x, pos->y, weights, dim, &sums);

    average->x = sums.x / sums.z;
    average->y = sums.y / sums.z;

    return seconds;
}
static float WeightedAverageSqrdSubGPU(const Floats2* const pos, const float* const weights,
    const int dim, float meanX, float meanY, Float2* const average) {

    ulong arrayBytes = dim * sizeof(float);

    float3 sums;
    sums.x = 0.0f;
    sums.y = 0.0f;
    sums.z = 0.0f;

    float seconds = SumArrayWeightsSqrdSubGPU(pos->x, pos->y, weights, dim, meanX, meanY, &sums);

    average->x = sums.x / sums.z;
    average->y = sums.y / sums.z;

    return seconds;
}

static float* CumSum(const float* const arr_in, const int dim) {
    float* cumSumArr = (float*)malloc(dim * sizeof(float));

    cumSumArr[0] = arr_in[0];

    for (int i = 1; i < dim; i++) {
        cumSumArr[i] += cumSumArr[i - 1] + arr_in[i];
    }

    return cumSumArr;
}

/*///////////////////////////////////*/
/*            kernels.cu             */
/*///////////////////////////////////*/
#define SHARED_MEMORY_BANKS 32
#define LOG_MEM_BANKS 5

// There were two BCAO optimisations in the paper - this one is fastest
#define CONFLICT_FREE_OFFSET(n) ((n) >> LOG_MEM_BANKS)

__global__ void prescan_arbitrary(float* output, float* input, int n, int powerOfTwo)
{
    extern __shared__ float temp[];// allocated on invocation
    int threadID = threadIdx.x;

    int ai = threadID;
    int bi = threadID + (n / 2);
    int bankOffsetA = CONFLICT_FREE_OFFSET(ai);
    int bankOffsetB = CONFLICT_FREE_OFFSET(bi);


    if (threadID < n) {
        temp[ai + bankOffsetA] = input[ai];
        temp[bi + bankOffsetB] = input[bi];
    }
    else {
        temp[ai + bankOffsetA] = 0;
        temp[bi + bankOffsetB] = 0;
    }


    int offset = 1;
    for (int d = powerOfTwo >> 1; d > 0; d >>= 1) // build sum in place up the tree
    {
        __syncthreads();
        if (threadID < d)
        {
            int ai = offset * (2 * threadID + 1) - 1;
            int bi = offset * (2 * threadID + 2) - 1;
            ai += CONFLICT_FREE_OFFSET(ai);
            bi += CONFLICT_FREE_OFFSET(bi);

            temp[bi] += temp[ai];
        }
        offset *= 2;
    }

    if (threadID == 0) {
        temp[powerOfTwo - 1 + CONFLICT_FREE_OFFSET(powerOfTwo - 1)] = 0; // clear the last element
    }

    for (int d = 1; d < powerOfTwo; d *= 2) // traverse down tree & build scan
    {
        offset >>= 1;
        __syncthreads();
        if (threadID < d)
        {
            int ai = offset * (2 * threadID + 1) - 1;
            int bi = offset * (2 * threadID + 2) - 1;
            ai += CONFLICT_FREE_OFFSET(ai);
            bi += CONFLICT_FREE_OFFSET(bi);

            int t = temp[ai];
            temp[ai] = temp[bi];
            temp[bi] += t;
        }
    }
    __syncthreads();

    if (threadID < n) {
        output[ai] = temp[ai + bankOffsetA];
        output[bi] = temp[bi + bankOffsetB];
    }
}
__global__ void prescan_arbitrary_unoptimized(float* output, float* input, int n, int powerOfTwo) {
    extern __shared__ float temp[];// allocated on invocation
    int threadID = threadIdx.x;

    if (threadID < n) {
        temp[2 * threadID] = input[2 * threadID]; // load input into shared memory
        temp[2 * threadID + 1] = input[2 * threadID + 1];
    }
    else {
        temp[2 * threadID] = 0;
        temp[2 * threadID + 1] = 0;
    }


    int offset = 1;
    for (int d = powerOfTwo >> 1; d > 0; d >>= 1) // build sum in place up the tree
    {
        __syncthreads();
        if (threadID < d)
        {
            int ai = offset * (2 * threadID + 1) - 1;
            int bi = offset * (2 * threadID + 2) - 1;
            temp[bi] += temp[ai];
        }
        offset *= 2;
    }

    if (threadID == 0) { temp[powerOfTwo - 1] = 0; } // clear the last element

    for (int d = 1; d < powerOfTwo; d *= 2) // traverse down tree & build scan
    {
        offset >>= 1;
        __syncthreads();
        if (threadID < d)
        {
            int ai = offset * (2 * threadID + 1) - 1;
            int bi = offset * (2 * threadID + 2) - 1;
            int t = temp[ai];
            temp[ai] = temp[bi];
            temp[bi] += t;
        }
    }
    __syncthreads();

    if (threadID < n) {
        output[2 * threadID] = temp[2 * threadID]; // write results to device memory
        output[2 * threadID + 1] = temp[2 * threadID + 1];
    }
}
__global__ void prescan_large(float* output, float* input, int n, float* sums) {
    extern __shared__ float temp[];

    int blockID = blockIdx.x;
    int threadID = threadIdx.x;
    int blockOffset = blockID * n;

    int ai = threadID;
    int bi = threadID + (n / 2);
    int bankOffsetA = CONFLICT_FREE_OFFSET(ai);
    int bankOffsetB = CONFLICT_FREE_OFFSET(bi);
    temp[ai + bankOffsetA] = input[blockOffset + ai];
    temp[bi + bankOffsetB] = input[blockOffset + bi];

    int offset = 1;
    for (int d = n >> 1; d > 0; d >>= 1) // build sum in place up the tree
    {
        __syncthreads();
        if (threadID < d)
        {
            int ai = offset * (2 * threadID + 1) - 1;
            int bi = offset * (2 * threadID + 2) - 1;
            ai += CONFLICT_FREE_OFFSET(ai);
            bi += CONFLICT_FREE_OFFSET(bi);

            temp[bi] += temp[ai];
        }
        offset *= 2;
    }
    __syncthreads();


    if (threadID == 0) {
        sums[blockID] = temp[n - 1 + CONFLICT_FREE_OFFSET(n - 1)];
        temp[n - 1 + CONFLICT_FREE_OFFSET(n - 1)] = 0;
    }

    for (int d = 1; d < n; d *= 2) // traverse down tree & build scan
    {
        offset >>= 1;
        __syncthreads();
        if (threadID < d)
        {
            int ai = offset * (2 * threadID + 1) - 1;
            int bi = offset * (2 * threadID + 2) - 1;
            ai += CONFLICT_FREE_OFFSET(ai);
            bi += CONFLICT_FREE_OFFSET(bi);

            int t = temp[ai];
            temp[ai] = temp[bi];
            temp[bi] += t;
        }
    }
    __syncthreads();

    output[blockOffset + ai] = temp[ai + bankOffsetA];
    output[blockOffset + bi] = temp[bi + bankOffsetB];
}
__global__ void prescan_large_unoptimized(float* output, float* input, int n, float* sums) {
    int blockID = blockIdx.x;
    int threadID = threadIdx.x;
    int blockOffset = blockID * n;

    extern __shared__ float temp[];
    temp[2 * threadID] = input[blockOffset + (2 * threadID)];
    temp[2 * threadID + 1] = input[blockOffset + (2 * threadID) + 1];

    int offset = 1;
    for (int d = n >> 1; d > 0; d >>= 1) // build sum in place up the tree
    {
        __syncthreads();
        if (threadID < d)
        {
            int ai = offset * (2 * threadID + 1) - 1;
            int bi = offset * (2 * threadID + 2) - 1;
            temp[bi] += temp[ai];
        }
        offset *= 2;
    }
    __syncthreads();


    if (threadID == 0) {
        sums[blockID] = temp[n - 1];
        temp[n - 1] = 0;
    }

    for (int d = 1; d < n; d *= 2) // traverse down tree & build scan
    {
        offset >>= 1;
        __syncthreads();
        if (threadID < d)
        {
            int ai = offset * (2 * threadID + 1) - 1;
            int bi = offset * (2 * threadID + 2) - 1;
            int t = temp[ai];
            temp[ai] = temp[bi];
            temp[bi] += t;
        }
    }
    __syncthreads();

    output[blockOffset + (2 * threadID)] = temp[2 * threadID];
    output[blockOffset + (2 * threadID) + 1] = temp[2 * threadID + 1];
}
__global__ void add(float* output, int length, float* n) {
    int blockID = blockIdx.x;
    int threadID = threadIdx.x;
    int blockOffset = blockID * length;

    output[blockOffset + threadID] += n[blockID];
}
__global__ void add(float* output, int length, float* n1, float* n2) {
    int blockID = blockIdx.x;
    int threadID = threadIdx.x;
    int blockOffset = blockID * length;

    output[blockOffset + threadID] += n1[blockID] + n2[blockID];
}
bool isPowerOfTwo(int x) {
    return x && !(x & (x - 1));
}
int nextPowerOfTwo(int x) {
    int power = 1;
    while (power < x) {
        power *= 2;
    }
    return power;
}

void scanSmallDeviceArray(float* d_out, float* d_in, int length, bool bcao) {
    int powerOfTwo = nextPowerOfTwo(length);

    if (bcao) {
        prescan_arbitrary << <1, (length + 1) / 2, 2 * powerOfTwo * sizeof(float) >> > (d_out, d_in, length, powerOfTwo);
    }
    else {
        prescan_arbitrary_unoptimized << <1, (length + 1) / 2, 2 * powerOfTwo * sizeof(float) >> > (d_out, d_in, length, powerOfTwo);
    }
}
void scanLargeDeviceArray(float* d_out, float* d_in, int length, bool bcao) {
    int remainder = length % (ELEMENTS_PER_BLOCK);
    if (remainder == 0) {
        scanLargeEvenDeviceArray(d_out, d_in, length, bcao);
    }
    else {
        // perform a large scan on a compatible multiple of elements
        int lengthMultiple = length - remainder;
        scanLargeEvenDeviceArray(d_out, d_in, lengthMultiple, bcao);

        // scan the remaining elements and add the (inclusive) last element of the large scan to this
        float* startOfOutputArray = &(d_out[lengthMultiple]);
        scanSmallDeviceArray(startOfOutputArray, &(d_in[lengthMultiple]), remainder, bcao);

        add << <1, remainder >> > (startOfOutputArray, remainder, &(d_in[lengthMultiple - 1]), &(d_out[lengthMultiple - 1]));
    }
}
void scanLargeEvenDeviceArray(float* d_out, float* d_in, int length, bool bcao) {
    const int blocks = length / ELEMENTS_PER_BLOCK;
    const int sharedMemArraySize = ELEMENTS_PER_BLOCK * sizeof(float);

    float* d_sums, * d_incr;
    hipMalloc((void**)&d_sums, blocks * sizeof(float));
    hipMalloc((void**)&d_incr, blocks * sizeof(float));

    if (bcao) {
        prescan_large << <blocks, THREADS_PER_BLOCK, 2 * sharedMemArraySize >> > (d_out, d_in, ELEMENTS_PER_BLOCK, d_sums);
    }
    else {
        prescan_large_unoptimized << <blocks, THREADS_PER_BLOCK, 2 * sharedMemArraySize >> > (d_out, d_in, ELEMENTS_PER_BLOCK, d_sums);
    }

    const int sumsArrThreadsNeeded = (blocks + 1) / 2;
    if (sumsArrThreadsNeeded > THREADS_PER_BLOCK) {
        // perform a large scan on the sums arr
        scanLargeDeviceArray(d_incr, d_sums, blocks, bcao);
    }
    else {
        // only need one block to scan sums arr so can use small scan
        scanSmallDeviceArray(d_incr, d_sums, blocks, bcao);
    }

    add << <blocks, ELEMENTS_PER_BLOCK >> > (d_out, ELEMENTS_PER_BLOCK, d_incr);

    hipFree(d_sums);
    hipFree(d_incr);
}
float scan(float* output, float* input, int length, bool bcao) {
    float* d_out, * d_in;
    const int arraySize = length * sizeof(float);

    hipMalloc((void**)&d_out, arraySize);
    hipMalloc((void**)&d_in, arraySize);
    hipMemcpy(d_out, output, arraySize, hipMemcpyHostToDevice);
    hipMemcpy(d_in, input, arraySize, hipMemcpyHostToDevice);

    // start timer
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    if (length > ELEMENTS_PER_BLOCK) {
        scanLargeDeviceArray(d_out, d_in, length, bcao);
    }
    else {
        scanSmallDeviceArray(d_out, d_in, length, bcao);
    }

    // end timer
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float elapsedTime = 0;
    hipEventElapsedTime(&elapsedTime, start, stop);

    hipMemcpy(output, d_out, arraySize, hipMemcpyDeviceToHost);

    hipFree(d_out);
    hipFree(d_in);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return elapsedTime;
}
//static float CumSumGPU(const float* const arr_in, const int dim, float* const cumSumArr) {
//    float seconds = 0.0f;
//
//    for (int i = 0; i < dim; i++) {
//        seconds += SumArrayGPU(arr_in, i + 1, &cumSumArr[i]);  // + 1 for the size of the subArr
//    }
//
//    return seconds;
//}
//static float PrefixSumGPU(float*  arr_in, const int dim, float* const cumSumArr) {
static float CumSumGPU(float* arr_in, const int dim, float* const cumSumArr) {
    float seconds = 0.0f;

    // full scan with BCAO
    seconds = scan(cumSumArr, arr_in, N, true);

    return seconds;
}

// find the index, in the sorted array (ascending order), to insert the element preserving the order
static int SearchSorted(const float* const sortedArry, const float element, const int dim) {
    int index = dim;

    for (int i = 0; i < dim - 1; i++) {
        if (element <= sortedArry[i] && element > sortedArry[i + 1]) {
            index = i;
        }
    }

    return index;
}
__global__ void SearchSortedGPU(const float* const sortedArry, const int dim, const float element, int* const indexOut, const ulong resPosition) {
    uint tid = threadIdx.x;
    ulong idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= dim)
        return;

    if (element <= sortedArry[idx] && element > sortedArry[idx + 1]) {
        indexOut[resPosition] = sortedArry[idx];
    }
}

__global__ void SimpleResampleGPUKernel(
    const Particles* const particles, Particles* const particlesOut,
    const float* const sortedArry, const int dim, hiprandState* states, const float equalWeight) {
    uint tid = threadIdx.x;
    ulong idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= dim)
        return;

    hiprand_init(idx, 0, 0, &states[idx]);
    float element = hiprand_uniform(&states[idx]);

    //dim3 grid(gridDim.x);
    //dim3 block(blockDim.x);

    // I would like to use dynamic parallelism
    // but the sync device side with child is deprecated
    // SearchSortedGPU
    // Starts with indexFound = dim -1
    // So search sorted will not branch that much
    //SearchSortedGPU << <grid, block >> > (sortedArry, dim - 1, element, indexOut, idx);

    // Wait for the child kernel to be finished
    //hipDeviceSynchronize();  // deprecated, nice

    int index = 0;

    for (int i = 0; i < dim; i++) {
        if (element <= sortedArry[idx] && element > sortedArry[idx + 1]) {
            index = i;
        }
    }

    // When all the indexes are found
    __syncthreads();

    // resample according to indexes
    particlesOut->x[idx] = particles->x[index];
    particlesOut->y[idx] = particles->y[index];
    particlesOut->heading[idx] = particles->heading[index];
    particlesOut->weights[idx] = equalWeight;
}


__global__ void DivisionKernel(float* const dividend, const uint dividendDim, const float divisor) {
    uint tid = threadIdx.x;
    ulong idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= dividendDim)
        return;

    dividend[idx] /= divisor;
}
static float ParallelDivisionGPU(float* const dividend, const uint dividendDim, const float divisor) {
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float seconds = 0.0f;

    uint numBlocks = (dividendDim + BLOCKSIZE - 1) / BLOCKSIZE;

    long arrayBytes = dividendDim * sizeof(float);

    float* d_dividend;
    CHECK(hipMalloc((void**)&d_dividend, arrayBytes));
    CHECK(hipMemcpy(d_dividend, dividend, arrayBytes, hipMemcpyHostToDevice));

    hipEventRecord(start);

    DivisionKernel << <numBlocks, BLOCKSIZE >> > (d_dividend, dividendDim, divisor);

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    //CHECK(hipDeviceSynchronize());
    CHECK(hipGetLastError());

    float iterMilliseconds = 0;
    hipEventElapsedTime(&iterMilliseconds, start, stop);
    float iterSeconds = iterMilliseconds / 1000.0f;
    seconds += iterSeconds;

    hipEventDestroy(start);
    hipEventDestroy(stop);

    // memcopy D2H
    CHECK(hipMemcpy(dividend, d_dividend, arrayBytes, hipMemcpyDeviceToHost));

    hipFree(d_dividend);

    //CHECK(hipDeviceReset());

    return seconds;
}


static void PredictCPU(Particles* const p, const Float2* const u, const Float2* const std, const float dt) {
    //""" move according to control input u (heading change, velocity)
    //    with noise Q(std heading change, std velocity)`"""
    srand((unsigned int)time(NULL));   // Initialization, should only be called once.

    for (int i = 0; i < N; i++) {
        float r = ((float)rand() / (float)(RAND_MAX));      // rand Returns a pseudo-random integer between 0 and RAND_MAX.

        // update heading
        p->heading[i] += u->x + (r * std->x);
        p->heading[i] = fmodf(p->heading[i], PI2);

        float dist = (u->y * dt) + (r * std->y);

        // move in the(noisy) commanded direction
        p->x[i] += cos(p->heading[i]) * dist;
        p->y[i] += sin(p->heading[i]) * dist;
    }
}
__global__ void PredictGPUKernel(Particles* const D_in, hiprandState* const states, const Float2 u, const Float2 std, const float dt) {
    uint tid = threadIdx.x;
    ulong idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= N)
        return;

    hiprand_init(idx, 0, 0, &states[idx]);

    //# update heading
    float heading = D_in->heading[idx];
    heading += u.x + (hiprand_normal(&states[idx]) * std.x);
    heading = std::fmod(heading, 2 * PI);

    //# move in the (noisy) commanded direction
    float dist = (u.y * dt) + (hiprand_uniform(&states[idx]) * std.y);
    float pos_x = D_in->x[idx];
    float pos_y = D_in->y[idx];
    pos_x += std::cos(heading) * dist;
    pos_y += std::sin(heading) * dist;

    D_in->x[idx] = pos_x;
    D_in->y[idx] = pos_y;
    D_in->heading[idx] = heading;
}
static float PredictGPU(Particles** particles, const Float2* const u, const Float2* const std, const float dt) {
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    ulong particlesBytes = sizeof(Particles);

    Particles* d_particlesIn;
    CHECK(hipMalloc((void**)&d_particlesIn, particlesBytes));
    CHECK(hipMemcpy(d_particlesIn, *particles, particlesBytes, hipMemcpyHostToDevice));
    //CHECK(hipMemcpy(d_particlesIn, pp, particlesBytes, hipMemcpyHostToDevice));
    CHECK(hipGetLastError());

    uint numBlocks = (N + BLOCKSIZE - 1) / BLOCKSIZE;
    hiprandState* devStates;
    hipMalloc((void**)&devStates, N * sizeof(hiprandState));
    CHECK(hipGetLastError());

    hipEventRecord(start);

    //__global__ void PredictGPUKernel(Particles * D_in, Particles * C_out, hiprandState * states, float* u, float* std, float dt) {
    PredictGPUKernel << <numBlocks, BLOCKSIZE >> > (d_particlesIn, devStates, (*u), (*std), dt);

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    CHECK(hipMemcpy(*particles, d_particlesIn, particlesBytes, hipMemcpyDeviceToHost));
    CHECK(hipGetLastError());

    CHECK(hipFree(d_particlesIn));
    CHECK(hipFree(devStates));

    float milliseconds = 0.0f;
    hipEventElapsedTime(&milliseconds, start, stop);
    float sec = milliseconds / 1000.0f;

    hipEventDestroy(start);
    hipEventDestroy(stop);

    //CHECK(hipDeviceReset());

    return sec;
}

//def update(particles, weights, z, R, landmarks) :
//  for i, landmark in enumerate(landmarks) :
//      distance = np.linalg.norm(particles[:, 0 : 2] - landmark, axis = 1)
//      weights *= scipy.stats.norm(distance, R).pdf(z[i])
//
//  weights += 1.e-300      # avoid round - off to zero
//  weights /= sum(weights) # normalize
static void UpdateCPU(Particles* const p, const float const* z, const float R, const Floats2 const* landmarks, const int numberOfLandmarks) {
    int size = N;

    Floats2 distance;
    distance.x = (float*)malloc(size * sizeof(float));
    distance.y = (float*)malloc(size * sizeof(float));
    float* normPdfs = (float*)malloc(size * sizeof(float));
    float* distanceMagnitudes = (float*)calloc(size, sizeof(float));

    for (int i = 0; i < numberOfLandmarks; i++) {
        //  distance = np.linalg.norm(particles[:, 0 : 2] - landmark, axis = 1)
        memcpy(distance.x, p->x, size * sizeof(float));
        memcpy(distance.y, p->y, size * sizeof(float));

        for (int j = 0; j < size; j++) {    // particles[:, 0 : 2] - landmark
            distance.x[j] -= landmarks->x[i];
            distance.y[j] -= landmarks->y[i];
        }

        for (int j = 0; j < size; j++) {    // np.linalg.norm
            distanceMagnitudes[j] = Magnitude(distance.x[j], distance.y[j]);
        }

        //  weights *= scipy.stats.norm(distance, R).pdf(z[i])
        for (int j = 0; j < size; j++) { // scipy.stats.norm(distance, R).pdf(z[i])
            normPdfs[j] = normpdf(z[i], distanceMagnitudes[j], R);
        }
        for (int j = 0; j < size; j++) { // weights *=  // element wise multiplication
            p->weights[j] *= normPdfs[j];
        }
    }

    free(distanceMagnitudes);
    free(normPdfs);
    free(distance.x);
    free(distance.y);

    float sum = FLT_EPSILON;  // avoid round - off to zero
    for (int i = 0; i < size; i++) {
        sum += p->weights[i];
    }

    for (int i = 0; i < size; i++) {
        p->weights[i] /= sum; // normalize
    }
}
__global__ void UpdateGPUKernel(Particles* const p, float* distanceX, float* distanceY, const float const* z, const float R, const Float2 landmark) {
    uint tid = threadIdx.x;
    ulong idx = blockIdx.x * blockDim.x + threadIdx.x;

    // boundary check
    if (idx >= N)
        return;

    distanceX[idx] -= landmark.x;
    distanceY[idx] -= landmark.y;

    float distanceMagnitude = MagnitudeGPU(distanceX[idx], distanceY[idx]);

    float normPdf = normPdfGPU(z[idx], distanceMagnitude, R);
    //printf("\nidx: %d, normPdf: %f", idx, normPdf);

    p->weights[idx] *= normPdf;
}
static float UpdateGPU(Particles** const p, const float const* z, const float R, const Floats2 const* landmarks, const int numberOfLandmarks) {
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float seconds = 0.0f;

    ulong particlesBytes = sizeof(Particles);

    Particles* d_particles;
    CHECK(hipMalloc((void**)&d_particles, particlesBytes));
    CHECK(hipMemcpy(d_particles, *p, particlesBytes, hipMemcpyHostToDevice));
    CHECK(hipGetLastError());

    ulong arrayBytes = N * sizeof(float);
    ulong landmarkBytes = numberOfLandmarks * sizeof(float);

    float* d_distanceX, * d_distanceY;
    CHECK(hipMalloc((void**)&d_distanceX, arrayBytes));
    CHECK(hipMalloc((void**)&d_distanceY, arrayBytes));
    CHECK(hipGetLastError());

    float* d_z;
    CHECK(hipMalloc((void**)&d_z, landmarkBytes));
    CHECK(hipMemcpy(d_z, z, landmarkBytes, hipMemcpyHostToDevice));
    CHECK(hipGetLastError());

    uint numBlocks = (N + BLOCKSIZE - 1) / BLOCKSIZE;

    for (int i = 0; i < numberOfLandmarks; i++) {
        Float2 landmark;
        landmark.x = landmarks->x[i];
        landmark.y = landmarks->y[i];

        CHECK(hipMemcpy(d_distanceX, (*p)->x, arrayBytes, hipMemcpyHostToDevice));
        CHECK(hipMemcpy(d_distanceY, (*p)->y, arrayBytes, hipMemcpyHostToDevice));

        hipEventRecord(start);

        UpdateGPUKernel << <numBlocks, BLOCKSIZE >> > (d_particles, d_distanceX, d_distanceY, d_z, R, landmark);

        hipEventRecord(stop);
        hipEventSynchronize(stop);

        //CHECK(hipDeviceSynchronize());

        // Update weights
        CHECK(hipMemcpy(*p, d_particles, particlesBytes, hipMemcpyDeviceToHost));
        CHECK(hipGetLastError());

        float iterMilliseconds = 0;
        hipEventElapsedTime(&iterMilliseconds, start, stop);
        float iterSeconds = iterMilliseconds / 1000.0;
        seconds += iterSeconds;
    }

    CHECK(hipFree(d_particles));
    CHECK(hipFree(d_distanceX));
    CHECK(hipFree(d_distanceY));
    CHECK(hipFree(d_z));

    // Normalization
    float sum = FLT_EPSILON;  // avoid round - off to zero

    //hipEventRecord(start);

    seconds += SumArrayGPU((*p)->weights, N, &sum);

    seconds += ParallelDivisionGPU((*p)->weights, N, sum);

    //CHECK(hipDeviceReset());

    return seconds;
}

//def estimate(particles, weights) :
//    """returns mean and variance of the weighted particles"""
//
//    pos = particles[:, 0 : 2]
//    mean = np.average(pos, weights = weights, axis = 0)
//    var = np.average((pos - mean) * *2, weights = weights, axis = 0)
//    return mean, var
// returns mean and variance of the weighted particles
static void EstimateCPU(const Particles* const p, Float2* const mean_out, Float2* const var_out) {

    Floats2 pos;
    pos.x = (float*)malloc(N * sizeof(float));
    memcpy(pos.x, p->x, N * sizeof(float));
    pos.y = (float*)malloc(N * sizeof(float));
    memcpy(pos.y, p->y, N * sizeof(float));

    // mean = np.average(pos, weights = weights, axis = 0)
    (*mean_out) = WeightedAverage(&pos, p->weights, N);
    // var = np.average((pos - mean) **2, weights = weights, axis = 0)
    for (int i = 0; i < N; i++) {
        pos.x[i] = (pos.x[i] - mean_out->x) * (pos.x[i] - mean_out->x);
        pos.y[i] = (pos.y[i] - mean_out->y) * (pos.y[i] - mean_out->y);
    }

    (*var_out) = WeightedAverage(&pos, p->weights, N);

    free(pos.x);
    free(pos.y);
}
static float EstimateGPU(Particles** p, Float2* const mean_out, Float2* const var_out) {
    float seconds = 0.0f;

    Floats2 pos;
    pos.x = (float*)malloc(N * sizeof(float));
    memcpy(pos.x, (*p)->x, N * sizeof(float));
    pos.y = (float*)malloc(N * sizeof(float));
    memcpy(pos.y, (*p)->y, N * sizeof(float));

    seconds += WeightedAverageGPU(&pos, (*p)->weights, N, mean_out);

    seconds += WeightedAverageSqrdSubGPU(&pos, (*p)->weights, N, mean_out->x, mean_out->y, var_out);

    free(pos.x);
    free(pos.y);

    return seconds;
}

// We don't resample at every epoch. 
// For example, if you received no new measurements you have not received any information from which the resample can benefit. 
// We can determine when to resample by using something called the *effective N*, 
// which approximately measures the number of particles which meaningfully contribute to the probability distribution.
//def neff(weights) :
//    return 1. / np.sum(np.square(weights))
static float Neff(const float* const weights, const int dim) {
    float res = 0.0f;
    float sum = FLT_EPSILON;

    for (int i = 0; i < dim; i++) {
        float squaredWeight = weights[i] * weights[i];
        sum += squaredWeight;
    }

    res = 1.0f / sum;
    return res;
}
static float NeffGPU(const float* const weightsIn, const int dim, float* const neffOut) {
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float seconds = 0.0f;

    float res = 0.0f;
    float sum = FLT_EPSILON;

    uint numBlock = (dim + BLOCKSIZE - 1) / BLOCKSIZE;


    long blocksBytes = numBlock * sizeof(float);
    long arrayBytes = dim * sizeof(float);

    float* weightsOut, * d_weightsIn, * d_weightsOut;
    CHECK(hipMalloc((void**)&d_weightsIn, arrayBytes));
    CHECK(hipMemcpy(d_weightsIn, weightsIn, arrayBytes, hipMemcpyHostToDevice));
    CHECK(hipMalloc((void**)&d_weightsOut, blocksBytes));
    CHECK(hipMemset((void*)d_weightsOut, 0, blocksBytes));
    weightsOut = (float*)malloc(blocksBytes * sizeof(float));

    hipEventRecord(start);

    SumSquaredParReduce << <numBlock, BLOCKSIZE >> > (d_weightsIn, d_weightsOut, dim);

    CHECK(hipDeviceSynchronize());
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float iterMilliseconds = 0;
    hipEventElapsedTime(&iterMilliseconds, start, stop);
    float iterSeconds = iterMilliseconds / 1000.0f;
    seconds += iterSeconds;

    hipEventDestroy(start);
    hipEventDestroy(stop);

    //CHECK(hipDeviceSynchronize());
    CHECK(hipGetLastError());

    // memcopy D2H
    CHECK(hipMemcpy(weightsOut, d_weightsOut, blocksBytes, hipMemcpyDeviceToHost));

    // check result
    for (uint i = 0; i < numBlock; i++) {
        sum += weightsOut[i];
    }

    res = 1.0f / sum;

    hipFree(d_weightsOut);
    hipFree(d_weightsIn);
    free(weightsOut);

    //CHECK(hipDeviceReset());
    (*neffOut) = res;

    return seconds;
}

//def simple_resample(particles, weights) :
//    N = len(particles)
//    cumulative_sum = np.cumsum(weights)
//    cumulative_sum[-1] = 1. # avoid round - off error
//    indexes = np.searchsorted(cumulative_sum, random(N))  // Return random floats in the half-open interval [0.0, 1.0).
//
//    # resample according to indexes
//    particles[:] = particles[indexes]
//    weights.fill(1.0 / N)
static void SimpleResample(Particles* const p) {
    int dim = N;

    //    cumulative_sum = np.cumsum(weights)
    float* cumSum_arr = CumSum(p->weights, dim);

    // indexes = np.searchsorted(cumulative_sum, random(N))  // Return random floats in the half-open interval [0.0, 1.0).
    int* indexes = (int*)malloc(dim * sizeof(int));
    srand((unsigned int)time(NULL));   // Initialization, should only be called once.
    float r = 0.0f;
    for (int i = 0; i < dim; i++) {
        r = ((float)rand() / (float)(RAND_MAX));      // rand Returns a pseudo-random integer between 0 and RAND_MAX.
        indexes[i] = SearchSorted(cumSum_arr, r, dim);
    }

    //  # resample according to indexes
    //  particles[:] = particles[indexes]
    //  weights.fill(1.0 / N)
    float equalWeight = 1.0f / dim;
    for (int i = 0; i < dim; i++) {
        p->x[i] = p->x[indexes[i]];
        p->y[i] = p->y[indexes[i]];
        p->heading[i] = p->heading[indexes[i]];
        p->weights[i] = equalWeight;
    }

    free(indexes);
    free(cumSum_arr);
}
static float SimpleResampleGPU(Particles** const p) {
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float seconds = 0.0f;

    int dim = N;

    float* cumSum_arr = (float*)malloc(N * sizeof(float));

    seconds += CumSumGPU((*p)->weights, dim, cumSum_arr);
    printf("\n\t\t CumSumGPU: %9.4f sec", seconds);

    uint numBlocks = (dim + BLOCKSIZE - 1) / BLOCKSIZE;

    hiprandState* devStates;
    hipMalloc((void**)&devStates, N * sizeof(hiprandState));

    float equalWeight = 1.0f / dim;

    float* d_cumSum_arr;
    CHECK(hipMalloc((void**)&d_cumSum_arr, N * sizeof(float)));
    CHECK(hipMemcpy(d_cumSum_arr, cumSum_arr, N * sizeof(float), hipMemcpyHostToDevice));

    Particles* d_particles, * d_particlesOut;
    ulong particlesBytes = sizeof(Particles);
    CHECK(hipMalloc((void**)&d_particles, particlesBytes));
    CHECK(hipMemcpy(d_particles, *p, particlesBytes, hipMemcpyHostToDevice));
    CHECK(hipMalloc((void**)&d_particlesOut, particlesBytes));
    // it does not need to be initialized since we will overwrite the value of every element

    hipEventRecord(start);

    SimpleResampleGPUKernel << <numBlocks, BLOCKSIZE >> > (
        d_particles, d_particlesOut,
        d_cumSum_arr, dim,
        devStates, equalWeight);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    //hipDeviceSynchronize();
    CHECK(hipGetLastError());

    float iterMilliseconds = 0;
    hipEventElapsedTime(&iterMilliseconds, start, stop);
    float iterSeconds = iterMilliseconds / 1000.0f;
    seconds += iterSeconds;

    hipEventDestroy(start);
    hipEventDestroy(stop);

    CHECK(hipMemcpy(*p, d_particlesOut, sizeof(Particles), hipMemcpyDeviceToHost));
    CHECK(hipGetLastError());

    hipFree(d_cumSum_arr);
    hipFree(d_particles);
    hipFree(d_particlesOut);

    //CHECK(hipDeviceReset());

    return seconds;
}

/*
 *  Device function: block parallel reduction based on warp unrolling
 */
__device__ void blockWarpUnroll(float* thisBlock, int blockDim, uint tid) {
    // in-place reduction in global memory
    for (int stride = blockDim / 2; stride > 32; stride >>= 1) {
        if (tid < stride)
            thisBlock[tid] += thisBlock[tid + stride];

        // synchronize within threadblock
        __syncthreads();
    }

    // unrolling warp
    if (tid < 32) {
        volatile float* vmem = thisBlock;
        vmem[tid] += vmem[tid + 32];
        vmem[tid] += vmem[tid + 16];
        vmem[tid] += vmem[tid + 8];
        vmem[tid] += vmem[tid + 4];
        vmem[tid] += vmem[tid + 2];
        vmem[tid] += vmem[tid + 1];
    }
}

__global__ void Norm_BlockUnroll8(float* in, float* out, const float add, const ulong n) {
    uint tid = threadIdx.x;
    ulong idx = blockIdx.x * blockDim.x + threadIdx.x;

    // boundary check
    if (idx >= n)
        return;

    // convert global data pointer to the local pointer of this block
    float* thisBlock = in + blockIdx.x * blockDim.x * 8;

    float a = 0.0f;
    float temp = 0.0f;

    // unrolling 8 blocks
    if (idx + 7 * blockDim.x < n) {
        for (int i = 0; i < 8; i++) {
            temp = in[idx + i * blockDim.x];
            temp += add;
            a += temp * temp;
        }
        in[idx] = a;
    }

    __syncthreads();

    // block parall. reduction based on warp unrolling 
    blockWarpUnroll(thisBlock, blockDim.x, tid);

    // write result for this block to global mem
    if (tid == 0)
        out[blockIdx.x] = thisBlock[0];
}


void particleFilterGPU(Particles** p, const int iterations, const float sensorStdError) {
    clock_t startClock, stopClock;
    double timer;
    startClock = clock();

    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("device %d: %s \n", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    printf("%d", prop.major * 10 + prop.minor);

    printf(" - particleFilter GPU - \n");

    unsigned int dim = N;

    printf("number of particles: %d \n", dim);

    // Start
    float duration = 0.0f;

    Float2 u;
    Float2 std;
    float dt = 0.1f;

    Floats2 landmarks;
    int numberOfLandmarks = 4;
    landmarks.x = (float*)malloc(numberOfLandmarks * sizeof(float));
    landmarks.y = (float*)malloc(numberOfLandmarks * sizeof(float));

    landmarks.x[0] = 0.0f;
    landmarks.y[0] = 2.0f;

    landmarks.x[1] = 5.0f;
    landmarks.y[1] = 10.0f;

    landmarks.x[2] = 12.0f;
    landmarks.y[2] = 24.0f;

    landmarks.x[3] = 18.0f;
    landmarks.y[3] = 21.0f;

    Float2 robotPosition;
    robotPosition.x = 0.0f;
    robotPosition.y = 0.0f;

    Float2* xs;
    xs = (Float2*)malloc(iterations * sizeof(Float2));

    for (int i = 0; i < iterations; i++) {
        printf("\n\n Iteration: %d", i);

        // Diagonal movement
        robotPosition.x += 1.0f;
        robotPosition.y += 1.0f;

        srand((unsigned int)time(NULL));   // Initialization, should only be called once.
        float r = 0.0f;
        float* zs = (float*)malloc(numberOfLandmarks * sizeof(float));
        for (int j = 0; j < numberOfLandmarks; j++) {
            Float2 landmark;
            landmark.x = landmarks.x[j];
            landmark.y = landmarks.y[j];
            Float2 distanceRobotLandmark = Minus(&landmark, &robotPosition);
            float magnitudeDistance = Magnitude(distanceRobotLandmark);
            r = ((float)rand() / (float)(RAND_MAX));      // rand Returns a pseudo-random integer between 0 and RAND_MAX.
            zs[j] = magnitudeDistance + (r * sensorStdError);
        }

        float tempTime = PredictGPU(p, &u, &std, dt);
        duration += tempTime;
        printf("\n\t PredictGPU time: %9.4f sec", tempTime);

        tempTime = UpdateGPU(p, zs, sensorStdError, &landmarks, numberOfLandmarks);
        duration += tempTime;
        printf("\n\t UpdateGPU: %9.4f sec", tempTime);

        float neff;
        tempTime = NeffGPU((*p)->weights, N, &neff);
        duration += tempTime;
        printf("\n\t\t NeffGPU: %9.4f sec", tempTime);

        //if (neff < N / 2.0f) {
        if (1) { // Only to test Re-sample
            // re-sample
            tempTime = SimpleResampleGPU(p);
            duration += tempTime;
            printf("\n\t SimpleResampleGPU: %9.4f sec", tempTime);
        }

        Float2 var;
        Float2 mean;

        tempTime = EstimateGPU(p, &mean, &var);
        duration += tempTime;
        printf("\n\t EstimateGPU: %9.4f sec", tempTime);

        xs[i] = mean;

        free(zs);
    }

    // End
    printf("\n\n Total execution EVENT time: %9.4f sec \n\n", duration);

    stopClock = clock();
    timer = ((double)(stopClock - startClock)) / (double)CLOCKS_PER_SEC;
    printf("\n\n Total execution time: %9.4f sec \n\n", timer);

    // write to file the results
    FILE* fptr;

    // Open a file in append mode
    fptr = fopen("C:\\Users\\andre\\ANDREA\\UNI\\GPU-COMPUTING\\ParticleFilterCUDA\\particleFilterGPU.txt", "a");

    // Append some text to the file
    fprintf(fptr, "Total execution EVENT time: %9.4f sec", duration);

    // Close the file
    fclose(fptr);

    free(landmarks.x);
    free(landmarks.y);
    free(xs);

    CHECK(hipDeviceReset());
}

void particleFilterCPU(Particles* const p, const int iterations, const float sensorStdError) {
    clock_t start, stop;
    double timer;

    printf(" - particleFilter CPU - \n");

    unsigned int dim = N;

    printf("number of particles: %d \n", dim);

    start = clock();

    // Start

    Float2 u;
    Float2 std;
    float dt = 0.1f;

    Floats2 landmarks;
    int numberOfLandmarks = 4;
    landmarks.x = (float*)malloc(numberOfLandmarks * sizeof(float));
    landmarks.y = (float*)malloc(numberOfLandmarks * sizeof(float));

    landmarks.x[0] = -1.0f;
    landmarks.y[0] = 2.0f;

    landmarks.x[1] = 5.0f;
    landmarks.y[1] = 10.0f;

    landmarks.x[2] = 12.0f;
    landmarks.y[2] = 24.0f;

    landmarks.x[3] = 18.0f;
    landmarks.y[3] = 21.0f;

    Float2 robotPosition;
    robotPosition.x = 0.0f;
    robotPosition.y = 0.0f;

    Float2* xs;
    xs = (Float2*)malloc(iterations * sizeof(Float2));

    for (int i = 0; i < iterations; i++) {
        printf("Iteration: %d\n", i);

        // Diagonal movement
        robotPosition.x += 1.0f;
        robotPosition.y += 1.0f;

        srand((unsigned int)time(NULL));   // Initialization, should only be called once.
        float r = 0.0f;
        float* zs = (float*)malloc(numberOfLandmarks * sizeof(float));
        for (int j = 0; j < numberOfLandmarks; j++) {
            Float2 landmark;
            landmark.x = landmarks.x[j];
            landmark.y = landmarks.y[j];
            Float2 distanceRobotLandmark = Minus(&landmark, &robotPosition);
            float magnitudeDistance = Magnitude(distanceRobotLandmark);
            r = ((float)rand() / (float)(RAND_MAX));      // rand Returns a pseudo-random integer between 0 and RAND_MAX.
            zs[j] = magnitudeDistance + (r * sensorStdError);
        }

        PredictCPU(p, &u, &std, dt);
        printf("\tPredictCPU\n");

        UpdateCPU(p, zs, sensorStdError, &landmarks, numberOfLandmarks);
        printf("\tUpdateCPU\n");

        float neff = Neff((p)->weights, N);
        //if (neff < N / 2.0f) {
        if (1) { // Only to test Resample

            // resample
            SimpleResample(p);
            printf("\tSimpleResample\n");
        }

        Float2 var;
        Float2 mean;

        EstimateCPU(p, &mean, &var);
        printf("\tEstimateCPU\n");

        xs[i] = mean;

        free(zs);
    }

    // End

    stop = clock();
    timer = ((double)(stop - start)) / (double)CLOCKS_PER_SEC;
    printf("\n\n Total CPU execution time: %9.4f sec \n\n", timer);

    // write to file the results
    FILE* fptr;

    // Open a file in append mode
    fptr = fopen("C:\\Users\\andre\\ANDREA\\UNI\\GPU-COMPUTING\\ParticleFilterCUDA\\particleFilterCPU.txt", "a");

    // Append some text to the file
    fprintf(fptr, "Total CPU execution time: %9.4f sec", timer);

    // Close the file
    fclose(fptr);

    free(landmarks.x);
    free(landmarks.y);
    free(xs);
}


int main()
{
    Float2 xRange;
    xRange.x = MinX;
    xRange.y = MaxX;

    Float2 yRange;
    yRange.x = MinY;
    yRange.y = MaxY;

    Float2 headingRange;
    headingRange.x = MinHeading;
    headingRange.y = MaxHeading;

    Particles* p = nullptr;

    CreateAndRandomInitialize_Particles(&p, N, &xRange, &yRange, &headingRange);

    //particleFilterCPU(p, ITERAIONS, 0.1f);

    particleFilterGPU(&p, ITERAIONS, 0.1f);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    CHECK(hipDeviceReset());

    return 0;
}
