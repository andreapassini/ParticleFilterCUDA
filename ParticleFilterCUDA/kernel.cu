#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

// Not recommended by NVIDIA for unpredictable side effects but it works
//for __syncthreads() and CHECK
#ifndef __HIPCC__ 
#define __HIPCC__
#endif
#include <hip/device_functions.h>

#include <stdio.h>
#include <cmath>
#include <string>
#include <time.h>

#include "common.h"
#include "Particle.h"
#include "Float2.h"

#define PI 3.141592f
#define PI2 2.0f * 3.141592f

#define IDX(i,j,n) (i*n+j)
#define ABS(x,y) (x-y>=0?x-y:y-x)

#define DIM 10'000

#define BLOCKSIZE 1024  // block dim 1D
#define NUMBLOCKS 1024  // grid dim 1D 
#define N (NUMBLOCKS * BLOCKSIZE)

#define MinX 0.0f
#define MaxX 1000.0f

#define MinY 0.0f
#define MaxY 1000.0f

#define MinHeading 0.0f
#define MaxHeading 3.0f

__global__ void addKernel(int* c, const int* a, const int* b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

/*
 *  Block by block parallel implementation without divergence (interleaved schema)
 */
__global__ void SumParReduce(int* in, int* out, ulong n) {

    uint tid = threadIdx.x;
    ulong idx = blockIdx.x * blockDim.x + threadIdx.x;

    // boundary check
    if (idx >= n)
        return;

    // convert global data pointer to the local pointer of this block
    int* thisBlock = in + blockIdx.x * blockDim.x;

    // in-place reduction in global memory
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride)
            thisBlock[tid] += thisBlock[tid + stride];

        // synchronize within threadblock
        __syncthreads();
    }

    // write result for this block to global mem
    if (tid == 0)
        out[blockIdx.x] = thisBlock[0];
}

/*
 *  Block by block parallel implementation without divergence (interleaved schema)
 */
__global__ void SumSquaredParReduce(float* in, float* out, const ulong n) {
    uint tid = threadIdx.x;
    ulong idx = blockIdx.x * blockDim.x + threadIdx.x;

    // boundary check
    if (idx >= n)
        return;

    // convert global data pointer to the local pointer of this block
    float* thisBlock = in + blockIdx.x * blockDim.x;

    // in-place reduction in global memory
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride)
            thisBlock[tid] += (thisBlock[tid + stride] * thisBlock[tid + stride]);

        // synchronize within threadblock
        __syncthreads();
    }

    // write result for this block to global mem
    if (tid == 0)
        out[blockIdx.x] = thisBlock[0];

}

__global__ void GenerateParticles(Particles* D_in, Particles* C_out, hiprandState* states, float2 x_range, float2 y_range, float2 heading_range) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    hiprand_init(tid, 0, 0, &states[tid]);
    float pos_x = x_range.x + ((x_range.y - x_range.x) * hiprand_uniform(&states[tid])); // Lerp
    float pos_y = y_range.x + ((y_range.y - y_range.x) * hiprand_uniform(&states[tid]));
    float heading = heading_range.x + ((heading_range.y - heading_range.x) * hiprand_uniform(&states[tid]));
    heading = std::fmod(heading, 2 * PI);

    C_out->x[tid] = pos_x;
    C_out->y[tid] = pos_y;
    C_out->heading[tid] = heading;
}

//# def predict(particles, u, std, dt=1.):
//# """ move according to control input u (heading change, velocity)
//# with noise Q (std heading change, std velocity)`"""
__global__ void PredictGPUKernel(Particles* D_in, Particles* C_out, hiprandState* states, float* u, float* std, float dt) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    dt = 1.0f;

    hiprand_init(tid, 0, 0, &states[tid]);

    //# update heading
    float heading = D_in->heading[tid];
    heading += u[0] + (hiprand_normal(&states[tid]) * std[1]);
    heading = std::fmod(heading, 2 * PI);

    //# move in the (noisy) commanded direction
    float dist = (u[1] * dt) + (hiprand_uniform(&states[tid]) * std[1]);
    float pos_x = D_in->x[tid];
    float pos_y = D_in->y[tid];
    pos_x += std::cos(heading) * dist;
    pos_y += std::sin(heading) * dist;

    C_out->x[tid] = pos_x;
    C_out->y[tid] = pos_y;
    C_out->heading[tid] = heading;
}

static float SumArrayGPU(const float* const arrIn, const int dim) {
    float sum = 0.0f;

    int blockSize = 1024;            // block dim 1D
    //int numBlock = 1024 * 1024;      // grid dim 1D
    int numBlock = (dim / blockSize);
    if (dim % blockSize != 0) {
        numBlock += 1;
    }

    long blocksBytes = numBlock * sizeof(float);
    long arrayBytes = dim * sizeof(float);

    float* arrOut, * d_arrIn, * d_arrOut;
    CHECK(hipMalloc((void**)&d_arrIn, arrayBytes));
    CHECK(hipMemcpy(d_arrIn, arrIn, arrayBytes, hipMemcpyHostToDevice));
    CHECK(hipMalloc((void**)&d_arrOut, blocksBytes));
    CHECK(hipMemset((void*)d_arrOut, 0, blocksBytes));
    arrOut = (float*)malloc(blocksBytes * sizeof(float));

    SumParReduce << <numBlock, blockSize >> > (d_arrIn, d_arrOut, dim);
    CHECK(hipDeviceSynchronize());
    CHECK(hipGetLastError());

    // memcopy D2H
    CHECK(hipMemcpy(arrOut, d_arrOut, blocksBytes, hipMemcpyDeviceToHost));

    // check result
    for (uint i = 0; i < numBlock; i++) {
        sum += arrOut[i];
    }

    hipFree(d_arrOut);
    hipFree(d_arrIn);
    free(arrOut);

    CHECK(hipDeviceReset());

    return sum;
}

//  def normpdf(x, mu=0, sigma=1):
static float normpdf(const float x, const float mu = 0.0f, const float sigma = 1.0f) {
    //  u = float((x - mu) / abs(sigma))
    //  y = exp(-u * u / 2) / (sqrt(2 * pi) * abs(sigma))
    //  return y
    float u = (x - mu) / abs(sigma);
    float y = exp(-u * u / 2) / (sqrt(PI2) * abs(sigma));
    return y;
}

float sqrdMagnitude(const float* const X, const int dim) {
    float sqrdMag = FLT_EPSILON;
    for (int i = 0; i < dim; i++) {
        sqrdMag += X[i] * X[i];
    }
    return sqrdMag;
}

static float magnitudeXY(const Particles* const p) {
    float sqrdMagX = sqrdMagnitude(p->x, p->size);
    float sqrdMagY = sqrdMagnitude(p->y, p->size);
    float magnitude = sqrt(sqrdMagX + sqrdMagY);
    return magnitude;
}

static Float2 WeightedAverage(const Floats2* const pos, const float* const weights, const int dim) {
    Float2 avg;
    avg.x = 0.0f;
    avg.y = 0.0f;

    // numpy implementation: avg = sum(a * weights) / sum(weights)

    float sumWeights = FLT_EPSILON; // avoid div by 0
    Float2 sumPos;
    sumPos.x = 0.0f;
    sumPos.y = 0.0f;
    for (int i = 0; i < dim; i++) {
        sumWeights += weights[i];
        sumPos.x += pos->x[i] * weights[i];
        sumPos.y += pos->y[i] * weights[i];
    }

    avg.x = sumPos.x / sumWeights;
    avg.y = sumPos.y / sumWeights;

    return avg;
}

static float* CumSum(const float* const arr_in, const int dim) {
    float* cumSumArr = (float*)malloc(dim * sizeof(float));

    for (int i = 0; i < dim; i++) {
        for (int j = 0; j <= i; j++) {
            cumSumArr[i] += arr_in[j];
        }
    }

    return cumSumArr;
}

static float* CumSumGPU(const float* const arr_in, const int dim) {
    float* cumSumArr = (float*)malloc(dim * sizeof(float));

    for (int i = 0; i < dim; i++) {
        cumSumArr[i] = SumArrayGPU(arr_in, i + 1);  // + 1 for the size of the subArr
    }

    return cumSumArr;
}

// find the index, in the sorted array (ascending order), to insert the element preserving the order
static int SearchSorted(const float* const sortedArry, const float element, const int dim) {
    int index = dim;

    for (int i = 0; i < dim - 1; i++) {
        if (element <= sortedArry[i] && element > sortedArry[i + 1]) {
            index = i;
        }
    }

    return index;
}

static void PredictCPU(Particles* const p, const Float2* const u, const Float2* const std, const float dt) {
    //""" move according to control input u (heading change, velocity)
    //    with noise Q(std heading change, std velocity)`"""
    srand((unsigned int)time(NULL));   // Initialization, should only be called once.

    for (int i = 0; i < p->size; i++) {
        float r = ((float)rand() / (float)(RAND_MAX));      // rand Returns a pseudo-random integer between 0 and RAND_MAX.

        // update heading
        p->heading[i] += u->x + (r * std->x);
        p->heading[i] = fmodf(p->heading[i], PI2);

        float dist = (u->y * dt) + (r * std->y);

        // move in the(noisy) commanded direction
        p->x[i] += cos(p->heading[i]) * dist;
        p->y[i] += sin(p->heading[i]) * dist;

        //PrintParticle(p, i);
    }
}

static void PredictGPU(Particles* const p, const Float2* const u, const Float2* const std, const float dt) {
    //""" move according to control input u (heading change, velocity)
    //    with noise Q(std heading change, std velocity)`"""
    srand((unsigned int)time(NULL));   // Initialization, should only be called once.

    for (int i = 0; i < p->size; i++) {
        float r = ((float)rand() / (float)(RAND_MAX));      // rand Returns a pseudo-random integer between 0 and RAND_MAX.

        // update heading
        p->heading[i] += u->x + (r * std->x);
        p->heading[i] = fmodf(p->heading[i], 2.0f * PI);

        float dist = (u->y * dt) + (r * std->y);

        // move in the(noisy) commanded direction
        p->x[i] += cos(p->heading[i]) * dist;
        p->y[i] += sin(p->heading[i]) * dist;

        //PrintParticle(p, i);
    }
}

//def update(particles, weights, z, R, landmarks) :
//  for i, landmark in enumerate(landmarks) :
//      distance = np.linalg.norm(particles[:, 0 : 2] - landmark, axis = 1)
//      weights *= scipy.stats.norm(distance, R).pdf(z[i])
//
//  weights += 1.e-300      # avoid round - off to zero
//  weights /= sum(weights) # normalize
static void UpdateCPU(Particles* const p, const float const* z, const float R, const Floats2 const* landmarks, const int numberOfLandmarks) {
    int size = p->size;

    for (int i = 0; i < numberOfLandmarks; i++) {
        //  distance = np.linalg.norm(particles[:, 0 : 2] - landmark, axis = 1)
        Floats2 distance;
        distance.x = (float*)malloc(size * sizeof(float));
        memcpy(distance.x, p->x, size * sizeof(float));
        distance.y = (float*)malloc(size * sizeof(float));
        memcpy(distance.y, p->y, size * sizeof(float));

        for (int j = 0; j < size; j++) {    // particles[:, 0 : 2] - landmark
            distance.x[j] -= landmarks->x[i];
            distance.y[j] -= landmarks->y[i];
        }
        float* distanceMagnitudes = (float*)calloc(size, sizeof(float));
        for (int j = 0; j < size; j++) {    // np.linalg.norm
            distanceMagnitudes[j] = Magnitude(distance.x[j], distance.y[j]);
        }

        //  weights *= scipy.stats.norm(distance, R).pdf(z[i])
        float* normPdfs = (float*)malloc(size * sizeof(float));
        for (int j = 0; j < size; j++) { // scipy.stats.norm(distance, R).pdf(z[i])
            normPdfs[j] *= normpdf(z[i], distanceMagnitudes[j], R);;
        }
        for (int j = 0; j < size; j++) { // weights *=  // element wise multiplication
            p->weights[j] *= normPdfs[j];
        }

        free(distance.x);
        free(distance.y);
        free(distanceMagnitudes);
        free(normPdfs);
    }

    float sum = 0.0f;
    for (int i = 0; i < size; i++) {
        p->weights[i] += FLT_EPSILON; // avoid round - off to zero
        sum += p->weights[i];
    }

    for (int i = 0; i < size; i++) {
        p->weights[i] /= sum; // normalize
    }
}

static void UpdateGPU(Particles* const p, const float const* z, const float R, const Floats2 const* landmarks) {
    int size = p->size;

    for (int i = 0; i < size; i++) {

        //  distance = np.linalg.norm(particles[:, 0 : 2] - landmark, axis = 1)
        Floats2 distance;
        distance.x = (float*)malloc(size * sizeof(float));
        memcpy(distance.x, p->x, size * sizeof(float));
        distance.y = (float*)malloc(size * sizeof(float));
        memcpy(distance.y, p->y, size * sizeof(float));

        for (int j = 0; j < size; j++) {    // particles[:, 0 : 2] - landmark
            distance.x[j] -= landmarks->x[i];
            distance.y[j] -= landmarks->y[i];
        }
        float* distanceMagnitudes = (float*)calloc(size, sizeof(float));
        for (int j = 0; j < size; j++) {    // np.linalg.norm
            distanceMagnitudes[j] = Magnitude(distance.x[j], distance.y[j]);
        }

        //  weights *= scipy.stats.norm(distance, R).pdf(z[i])
        float* normPdfs = (float*)malloc(size * sizeof(float));
        for (int j = 0; j < size; j++) { // scipy.stats.norm(distance, R).pdf(z[i])
            normPdfs[j] *= normpdf(z[i], distanceMagnitudes[j], R);;
        }
        for (int j = 0; j < size; j++) { // weights *=  // element wise multiplication
            p->weights[j] *= normPdfs[j];
        }

        free(distance.x);
        free(distance.y);
        free(distanceMagnitudes);
        free(normPdfs);
    }

    float sum = 0.0f;
    for (int i = 0; i < size; i++) {
        p->weights[i] += FLT_EPSILON; // avoid round - off to zero
        sum += p->weights[i];
    }

    for (int i = 0; i < size; i++) {
        p->weights[i] /= sum; // normalize
    }
}

//def estimate(particles, weights) :
//    """returns mean and variance of the weighted particles"""
//
//    pos = particles[:, 0 : 2]
//    mean = np.average(pos, weights = weights, axis = 0)
//    var = np.average((pos - mean) * *2, weights = weights, axis = 0)
//    return mean, var
// returns mean and variance of the weighted particles
static void EstimateCPU(const Particles* const p, Float2* const mean_out, Float2* const var_out) {

    Floats2 pos;
    pos.x = (float*)malloc(p->size * sizeof(float));
    memcpy(pos.x, p->x, p->size * sizeof(float));
    pos.y = (float*)malloc(p->size * sizeof(float));
    memcpy(pos.y, p->y, p->size * sizeof(float));

    // mean = np.average(pos, weights = weights, axis = 0)
    (*mean_out) = WeightedAverage(&pos, p->weights, p->size);
    // var = np.average((pos - mean) **2, weights = weights, axis = 0)
    for (int i = 0; i < p->size; i++) {
        pos.x[i] = (pos.x[i] - mean_out->x) * (pos.x[i] - mean_out->x);
        pos.y[i] = (pos.y[i] - mean_out->y) * (pos.y[i] - mean_out->y);
    }

    (*var_out) = WeightedAverage(&pos, p->weights, p->size);

    free(pos.x);
    free(pos.y);
}

//def simple_resample(particles, weights) :
//    N = len(particles)
//    cumulative_sum = np.cumsum(weights)
//    cumulative_sum[-1] = 1. # avoid round - off error
//    indexes = np.searchsorted(cumulative_sum, random(N))  // Return random floats in the half-open interval [0.0, 1.0).
//
//    # resample according to indexes
//    particles[:] = particles[indexes]
//    weights.fill(1.0 / N)
static void SimpleResample(Particles* const p) {
    int dim = p->size;

    //    cumulative_sum = np.cumsum(weights)
    float* cumSum_arr = CumSum(p->weights, dim);

    // indexes = np.searchsorted(cumulative_sum, random(N))  // Return random floats in the half-open interval [0.0, 1.0).
    int* indexes = (int*)malloc(dim * sizeof(int));
    srand((unsigned int)time(NULL));   // Initialization, should only be called once.
    float r = 0.0f;
    for (int i = 0; i < dim; i++) {
        r = ((float)rand() / (float)(RAND_MAX));      // rand Returns a pseudo-random integer between 0 and RAND_MAX.
        indexes[i] = SearchSorted(cumSum_arr, r, dim);
    }

    //  # resample according to indexes
    //  particles[:] = particles[indexes]
    //  weights.fill(1.0 / N)
    float equalWeight = 1.0f / dim;
    for (int i = 0; i < dim; i++) {
        p->x[i] = p->x[indexes[i]];
        p->y[i] = p->y[indexes[i]];
        p->heading[i] = p->heading[indexes[i]];
        p->weights[i] = equalWeight;
    }

    free(indexes);
    free(cumSum_arr);
}

// We don't resample at every epoch. 
// For example, if you received no new measurements you have not received any information from which the resample can benefit. 
// We can determine when to resample by using something called the *effective N*, 
// which approximately measures the number of particles which meaningfully contribute to the probability distribution.
//def neff(weights) :
//    return 1. / np.sum(np.square(weights))
static float Neff(const float* const weights, const int dim) {
    float res = 0.0f;
    float sum = FLT_EPSILON;

    for (int i = 0; i < dim; i++) {
        float squaredWeight = weights[i] * weights[i];
        sum += squaredWeight;
    }

    res = 1.0f / sum;
    return res;
}

static float NeffGPU(const float* const weightsIn, const int dim) {
    float res = 0.0f;
    float sum = FLT_EPSILON;

    int blockSize = 1024;            // block dim 1D
    //int numBlock = 1024 * 1024;      // grid dim 1D
    int numBlock = (dim / blockSize);
    if (dim % blockSize != 0) {
        numBlock += 1;
    }

    long blocksBytes = numBlock * sizeof(float);
    long arrayBytes = dim * sizeof(float);

    float* weightsOut, * d_weightsIn, * d_weightsOut;
    CHECK(hipMalloc((void**)&d_weightsIn, arrayBytes));
    CHECK(hipMemcpy(d_weightsIn, weightsIn, arrayBytes, hipMemcpyHostToDevice));
    CHECK(hipMalloc((void**)&d_weightsOut, blocksBytes));
    CHECK(hipMemset((void*)d_weightsOut, 0, blocksBytes));
    weightsOut = (float*)malloc(blocksBytes * sizeof(float));

    SumSquaredParReduce << <numBlock, blockSize >> > (d_weightsIn, d_weightsOut, dim);
    CHECK(hipDeviceSynchronize());
    CHECK(hipGetLastError());

    // memcopy D2H
    CHECK(hipMemcpy(weightsOut, d_weightsOut, blocksBytes, hipMemcpyDeviceToHost));

    // check result
    for (uint i = 0; i < numBlock; i++) {
        sum += weightsOut[i];
    }

    res = 1.0f / sum;

    hipFree(d_weightsOut);
    hipFree(d_weightsIn);
    free(weightsOut);

    CHECK(hipDeviceReset());

    return res;
}

/*
 *  Device function: block parallel reduction based on warp unrolling
 */
__device__ void blockWarpUnroll(float* thisBlock, int blockDim, uint tid) {
    // in-place reduction in global memory
    for (int stride = blockDim / 2; stride > 32; stride >>= 1) {
        if (tid < stride)
            thisBlock[tid] += thisBlock[tid + stride];

        // synchronize within threadblock
        __syncthreads();
    }

    // unrolling warp
    if (tid < 32) {
        volatile float* vmem = thisBlock;
        vmem[tid] += vmem[tid + 32];
        vmem[tid] += vmem[tid + 16];
        vmem[tid] += vmem[tid + 8];
        vmem[tid] += vmem[tid + 4];
        vmem[tid] += vmem[tid + 2];
        vmem[tid] += vmem[tid + 1];
    }
}

__global__ void Norm_BlockUnroll8(float* in, float* out, float add, ulong n) {
    uint tid = threadIdx.x;
    ulong idx = blockIdx.x * blockDim.x + threadIdx.x;

    // boundary check
    if (idx >= n)
        return;

    // convert global data pointer to the local pointer of this block
    float* thisBlock = in + blockIdx.x * blockDim.x * 8;

    float a = 0.0f;
    float temp = 0.0f;

    // unrolling 8 blocks
    if (idx + 7 * blockDim.x < n) {
        for (int i = 0; i < 8; i++) {
            temp = in[idx + i * blockDim.x];
            temp += add;
            a += temp * temp;
        }
        in[idx] = a;
    }

    __syncthreads();

    // block parall. reduction based on warp unrolling 
    blockWarpUnroll(thisBlock, blockDim.x, tid);

    // write result for this block to global mem
    if (tid == 0)
        out[blockIdx.x] = thisBlock[0];
}

//# def update(particles, weights, z, R, landmarks):
__global__ void Update(float2 norm, Particles* particles, Particles* C_out, float* weights, float* z, float* landmarks, int numberOfLandmarks, float R) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // boundary check
    if (tid >= DIM)
        return;

    //  distance = np.linalg.norm(particles[:, 0 : 2] - landmark, axis = 1)
    //  weights *= scipy.stats.norm(distance, R).pdf(z[i])

}


void particleFilterGPU(Particles* p) {

    Particles d_p;

    CHECK(hipMalloc((void**)&d_p.x, DIM * sizeof(float)));
    CHECK(hipMemcpy(d_p.x, p->x, DIM * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMalloc((void**)&d_p.y, DIM * sizeof(float)));
    CHECK(hipMemcpy(d_p.y, p->y, DIM * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMalloc((void**)&d_p.weights, DIM * sizeof(float)));
    CHECK(hipMemcpy(d_p.weights, p->weights, DIM * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMalloc((void**)&d_p.heading, DIM * sizeof(float)));
    CHECK(hipMemcpy(d_p.heading, p->heading, DIM * sizeof(float), hipMemcpyHostToDevice));

    // OUTPUT for Device
    Particles d_out;
    CHECK(hipMalloc((void**)&d_out.x, DIM * sizeof(float)));
    CHECK(hipMemcpy(d_out.x, p->x, DIM * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMalloc((void**)&d_out.y, DIM * sizeof(float)));
    CHECK(hipMemcpy(d_out.y, p->y, DIM * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMalloc((void**)&d_out.weights, DIM * sizeof(float)));
    CHECK(hipMemcpy(d_out.weights, p->weights, DIM * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMalloc((void**)&d_out.heading, DIM * sizeof(float)));
    CHECK(hipMemcpy(d_out.heading, p->heading, DIM * sizeof(float), hipMemcpyHostToDevice));


    hipFree(d_p.x);
    hipFree(d_p.y);
    hipFree(d_p.heading);
    hipFree(d_p.weights);
    hipFree(d_out.x);
    hipFree(d_out.y);
    hipFree(d_out.heading);
    hipFree(d_out.weights);
}

void particleFilterCPU(Particles* const p, const int iterations, const float sensorStdError) {
    clock_t start, stop;
    double timer;

    printf(" - particleFilterC - \n");

    unsigned int dim = p->size;

    printf("number of particles: %d \n", dim);

    start = clock();

    // Start

    Float2 u;
    Float2 std;
    float dt = 0.1f;

    Floats2 landmarks;
    int numberOfLandmarks = 4;
    landmarks.x = (float*)malloc(numberOfLandmarks * sizeof(float));
    landmarks.y = (float*)malloc(numberOfLandmarks * sizeof(float));

    landmarks.x[0] = -1.0f;
    landmarks.y[0] = 2.0f;

    landmarks.x[1] = 5.0f;
    landmarks.y[1] = 10.0f;

    landmarks.x[2] = 12.0f;
    landmarks.y[2] = 24.0f;

    landmarks.x[3] = 18.0f;
    landmarks.y[3] = 21.0f;

    Float2 robotPosition;
    robotPosition.x = 0.0f;
    robotPosition.y = 0.0f;

    Float2* xs;
    xs = (Float2*)malloc(iterations * sizeof(Float2));

    for (int i = 0; i < iterations; i++) {
        // Diagonal movement
        robotPosition.x += 1.0f;
        robotPosition.y += 1.0f;

        srand((unsigned int)time(NULL));   // Initialization, should only be called once.
        float r = 0.0f;
        float* zs = (float*)malloc(numberOfLandmarks * sizeof(float));
        for (int j = 0; j < numberOfLandmarks; j++) {
            Float2 landmark;
            landmark.x = landmarks.x[j];
            landmark.y = landmarks.y[j];
            Float2 distanceRobotLandmark = Minus(&landmark, &robotPosition);
            float magnitudeDistance = Magnitude(distanceRobotLandmark);
            r = ((float)rand() / (float)(RAND_MAX));      // rand Returns a pseudo-random integer between 0 and RAND_MAX.
            zs[j] = magnitudeDistance + (r * sensorStdError);
        }

        PredictCPU(p, &u, &std, dt);

        UpdateCPU(p, zs, sensorStdError, &landmarks, numberOfLandmarks);

        //# resample if too few effective particles
        //    if neff(weights) < N / 2:
        //indexes = systematic_resample(weights)
        //    resample_from_index(particles, weights, indexes)
        //    assert np.allclose(weights, 1 / N)
        //mu, var = estimate(particles, weights)
        //xs.append(mu)

        float neff = Neff(p->weights, p->size);
        if (neff < p->size / 2.0f) {
            // resample
            SimpleResample(p);
        }

        Float2 var;
        Float2 mean;

        EstimateCPU(p, &mean, &var);

        xs[i] = mean;

        free(zs);
    }

    // End

    stop = clock();
    timer = ((double)(stop - start)) / (double)CLOCKS_PER_SEC;
    printf("\n\n Total execution time: %9.4f sec \n\n", timer);

    free(landmarks.x);
    free(landmarks.y);
    free(xs);
}

void euclideanNorm(Particles* p, float2* norm, float2* landmark) {

    Particles d_p;

    CHECK(hipMalloc((void**)&d_p.x, DIM * sizeof(float)));
    CHECK(hipMemcpy(d_p.x, p->x, DIM * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMalloc((void**)&d_p.y, DIM * sizeof(float)));
    CHECK(hipMemcpy(d_p.y, p->y, DIM * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMalloc((void**)&d_p.weights, DIM * sizeof(float)));
    CHECK(hipMemcpy(d_p.weights, p->weights, DIM * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMalloc((void**)&d_p.heading, DIM * sizeof(float)));
    CHECK(hipMemcpy(d_p.heading, p->heading, DIM * sizeof(float), hipMemcpyHostToDevice));


    // OUTPUT for Device
    Particles d_out;
    CHECK(hipMalloc((void**)&d_out.x, DIM * sizeof(float)));
    CHECK(hipMemcpy(d_out.x, p->x, DIM * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMalloc((void**)&d_out.y, DIM * sizeof(float)));
    CHECK(hipMemcpy(d_out.y, p->y, DIM * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMalloc((void**)&d_out.weights, DIM * sizeof(float)));
    CHECK(hipMemcpy(d_out.weights, p->weights, DIM * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMalloc((void**)&d_out.heading, DIM * sizeof(float)));
    CHECK(hipMemcpy(d_out.heading, p->heading, DIM * sizeof(float), hipMemcpyHostToDevice));


    Particles p_norm;

    Create_Particles(&p_norm, DIM);

    Norm_BlockUnroll8 << <NUMBLOCKS / 8, BLOCKSIZE >> > (d_p.x, d_out.x, -landmark->x, N);  // ERRROR at <<< can be simply ignored
    Norm_BlockUnroll8 << <NUMBLOCKS / 8, BLOCKSIZE >> > (d_p.y, d_out.y, -landmark->y, N);

    CHECK(hipDeviceSynchronize());
    CHECK(hipGetLastError());

    CHECK(hipMemcpy(p_norm.x, d_out.x, DIM * sizeof(float), hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(p_norm.y, d_out.y, DIM * sizeof(float), hipMemcpyDeviceToHost));

    // Sum on CPU the last elements
    for (uint i = 0; i < NUMBLOCKS / 8; i++) {
        norm->x += p_norm.x[i];
        norm->y += p_norm.y[i];
    }

    hipFree(d_p.x);
    hipFree(d_p.y);
    hipFree(d_p.heading);
    hipFree(d_p.weights);
    hipFree(d_out.x);
    hipFree(d_out.y);
    hipFree(d_out.heading);
    hipFree(d_out.weights);
    free(p_norm.x);
    free(p_norm.y);
    free(p_norm.heading);
    free(p_norm.weights);
}

int main()
{
    Float2 xRange;
    xRange.x = MinX;
    xRange.y = MaxX;

    Float2 yRange;
    yRange.x = MinY;
    yRange.y = MaxY;

    Float2 headingRange;
    headingRange.x = MinHeading;
    headingRange.y = MaxHeading;

    Particles p;

    CreateAndRandomInitialize_Particles(&p, DIM, &xRange, &yRange, &headingRange);

    //particleFilterGPU(&p);

    particleFilterCPU(&p, 18, 0.1f);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    CHECK(hipDeviceReset());

    return 0;
}


//// Helper function for using CUDA to add vectors in parallel.
//hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
//{
//    int *dev_a = 0;
//    int *dev_b = 0;
//    int *dev_c = 0;
//    hipError_t cudaStatus;
//
//    //// Choose which GPU to run on, change this on a multi-GPU system.
//    cudaStatus = hipSetDevice(0);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
//        goto Error;
//    }
//
//    // Allocate GPU buffers for three vectors (two input, one output)    .
//    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMalloc failed!");
//        goto Error;
//    }
//
//    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMalloc failed!");
//        goto Error;
//    }
//
//    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMalloc failed!");
//        goto Error;
//    }
//
//    // Copy input vectors from host memory to GPU buffers.
//    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMemcpy failed!");
//        goto Error;
//    }
//
//    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMemcpy failed!");
//        goto Error;
//    }
//
//    // Launch a kernel on the GPU with one thread for each element.
//    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);
//
//    // Check for any errors launching the kernel
//    cudaStatus = hipGetLastError();
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
//        goto Error;
//    }
//    
//    // hipDeviceSynchronize waits for the kernel to finish, and returns
//    // any errors encountered during the launch.
//    cudaStatus = hipDeviceSynchronize();
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
//        goto Error;
//    }
//
//    // Copy output vector from GPU buffer to host memory.
//    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMemcpy failed!");
//        goto Error;
//    }
//
//Error:
//    hipFree(dev_c);
//    hipFree(dev_a);
//    hipFree(dev_b);
//    
//    return cudaStatus;
//}

