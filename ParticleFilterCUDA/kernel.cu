#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#include "common.h"
#include "Particle.h"

#include <stdio.h>
#include <cmath>

#define PI 3.141592f

#define IDX(i,j,n) (i*n+j)
#define ABS(x,y) (x-y>=0?x-y:y-x)
#define DIM 100


hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);



__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

__global__ void GenerateParticles(Particle* D_in, Particle* C_out, hiprandState* states, Vec2 x_range, Vec2 y_range, Vec2 heading_range) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    hiprand_init(tid, 0, 0, &states[tid]);

    D_in->x = x_range.x + ((x_range.y - x_range.x) * hiprand_uniform(&states[tid]));

    p.position.x = x_range.x + ((x_range.y - x_range.x) * hiprand_uniform(&states[tid]));
    p.position.y = y_range.x + ((y_range.y - y_range.x) * hiprand_uniform(&states[tid]));
    p.heading = hiprand_uniform(&states[tid]);
    p.heading = std::fmod(p.heading, 2 * PI);

    C_out[tid] = p;
}

int main()
{
    Particle p;
    Particle d_p;

    CreateParticleDim(&p, DIM);

    CHECK(hipMalloc((void**)&d_p.x, DIM * sizeof(float)));
    CHECK(hipMemcpy(d_p.x, p.x, DIM*sizeof(float), hipMemcpyHostToDevice));

    CHECK(hipMalloc((void**)&d_p.y, DIM * sizeof(float)));
    CHECK(hipMemcpy(d_p.y, p.y, DIM * sizeof(float), hipMemcpyHostToDevice));

    CHECK(hipMalloc((void**)&d_p.weight, DIM * sizeof(float)));
    CHECK(hipMemcpy(d_p.weight, p.weight, DIM * sizeof(float), hipMemcpyHostToDevice));

    CHECK(hipMalloc((void**)&d_p.heading, DIM * sizeof(float)));
    CHECK(hipMemcpy(d_p.heading, p.heading, DIM * sizeof(float), hipMemcpyHostToDevice));


    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    Particle p;

    p.x = (float*)malloc(DIM * sizeof(Particle));

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    //// Choose which GPU to run on, change this on a multi-GPU system.
    //cudaStatus = hipSetDevice(0);
    //if (cudaStatus != hipSuccess) {
    //    fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
    //    goto Error;
    //}

    //// Allocate GPU buffers for three vectors (two input, one output)    .
    //cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    //if (cudaStatus != hipSuccess) {
    //    fprintf(stderr, "hipMalloc failed!");
    //    goto Error;
    //}

    //cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    //if (cudaStatus != hipSuccess) {
    //    fprintf(stderr, "hipMalloc failed!");
    //    goto Error;
    //}

    //cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    //if (cudaStatus != hipSuccess) {
    //    fprintf(stderr, "hipMalloc failed!");
    //    goto Error;
    //}

    //// Copy input vectors from host memory to GPU buffers.
    //cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    //if (cudaStatus != hipSuccess) {
    //    fprintf(stderr, "hipMemcpy failed!");
    //    goto Error;
    //}

    //cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    //if (cudaStatus != hipSuccess) {
    //    fprintf(stderr, "hipMemcpy failed!");
    //    goto Error;
    //}

    //// Launch a kernel on the GPU with one thread for each element.
    //addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    //// Check for any errors launching the kernel
    //cudaStatus = hipGetLastError();
    //if (cudaStatus != hipSuccess) {
    //    fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
    //    goto Error;
    //}
    //
    //// hipDeviceSynchronize waits for the kernel to finish, and returns
    //// any errors encountered during the launch.
    //cudaStatus = hipDeviceSynchronize();
    //if (cudaStatus != hipSuccess) {
    //    fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
    //    goto Error;
    //}

    //// Copy output vector from GPU buffer to host memory.
    //cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    //if (cudaStatus != hipSuccess) {
    //    fprintf(stderr, "hipMemcpy failed!");
    //    goto Error;
    //}

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
