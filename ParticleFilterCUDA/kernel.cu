#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

// Not recommended by NVIDIA for unpredictable side effects but it works
//for __syncthreads() and CHECK
#ifndef __HIPCC__ 
#define __HIPCC__
#endif
#include <hip/device_functions.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>

#include <stdio.h>
#include <cmath>
#include <string>
#include <time.h>

#include "common.h"
#include "Particle.h"
#include "Float2.h"

#define PI 3.141592f
#define PI2 2.0f * PI

#define IDX(i,j,n) (i*n+j)
#define ABS(x,y) (x-y>=0?x-y:y-x)

#define DIM 1'000'000

#define BLOCKSIZE 1024  // block dim 1D
#define NUMBLOCKS 1024  // grid dim 1D 
#define N (NUMBLOCKS * BLOCKSIZE)

#define MinX 0.0f
#define MaxX 1000.0f

#define MinY 0.0f
#define MaxY 1000.0f

#define MinHeading 0.0f
#define MaxHeading 3.0f

__global__ void addKernel(int* c, const int* a, const int* b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

/*
 *  Block by block parallel implementation without divergence (interleaved schema)
 */
__global__ void SumParReduce(float* in, float* out, ulong n) {

    uint tid = threadIdx.x;
    ulong idx = blockIdx.x * blockDim.x + threadIdx.x;

    // boundary check
    if (idx >= n)
        return;

    // convert global data pointer to the local pointer of this block
    float* thisBlock = in + blockIdx.x * blockDim.x;

    // in-place reduction in global memory
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride)
            thisBlock[tid] += thisBlock[tid + stride];

        // synchronize within threadblock
        __syncthreads();
    }

    // write result for this block to global mem
    if (tid == 0)
        out[blockIdx.x] = thisBlock[0];
}

/*
 *  Block by block parallel implementation without divergence (interleaved schema)
 */
__global__ void SumSquaredParReduce(float* in, float* out, const ulong n) {
    uint tid = threadIdx.x;
    ulong idx = blockIdx.x * blockDim.x + threadIdx.x;

    // boundary check
    if (idx >= n)
        return;

    // convert global data pointer to the local pointer of this block
    float* thisBlock = in + blockIdx.x * blockDim.x;

    // in-place reduction in global memory
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride)
            thisBlock[tid] += (thisBlock[tid + stride] * thisBlock[tid + stride]);

        // synchronize within threadblock
        __syncthreads();
    }

    // write result for this block to global mem
    if (tid == 0)
        out[blockIdx.x] = thisBlock[0];
}

__global__ void GenerateParticles(Particles* D_in, Particles* C_out, hiprandState* states, float2 x_range, float2 y_range, float2 heading_range) {
    uint tid = threadIdx.x;
    ulong idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= D_in->size)
        return;

    hiprand_init(idx, 0, 0, &states[idx]);
    float pos_x = x_range.x + ((x_range.y - x_range.x) * hiprand_uniform(&states[idx]));
    float pos_y = y_range.x + ((y_range.y - y_range.x) * hiprand_uniform(&states[idx]));
    float heading = heading_range.x + ((heading_range.y - heading_range.x) * hiprand_uniform(&states[idx]));
    heading = std::fmod(heading, 2 * PI);

    C_out->x[idx] = pos_x;
    C_out->y[idx] = pos_y;
    C_out->heading[idx] = heading;
}

__global__ void PredictGPUKernel(Particles* const D_in, Particles* const C_out, hiprandState* const states, const Float2 u, const Float2 std, const float dt) {
    uint tid = threadIdx.x;
    ulong idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= D_in->size)
        return;

    hiprand_init(idx, 0, 0, &states[idx]);

    //# update heading
    float heading = D_in->heading[idx];
    heading += u.x + (hiprand_normal(&states[idx]) * std.x);
    heading = std::fmod(heading, 2 * PI);

    //# move in the (noisy) commanded direction
    float dist = (u.y * dt) + (hiprand_uniform(&states[idx]) * std.y);
    float pos_x = D_in->x[idx];
    float pos_y = D_in->y[idx];
    pos_x += std::cos(heading) * dist;
    pos_y += std::sin(heading) * dist;

    C_out->x[idx] = pos_x;
    C_out->y[idx] = pos_y;
    C_out->heading[idx] = heading;
}

static float SumArrayGPU(const float* const arrIn, const int dim) {
    float sum = FLT_EPSILON;

    uint numBlocks = (dim + BLOCKSIZE - 1) / BLOCKSIZE;

    long blocksBytes = numBlocks * sizeof(float);
    long arrayBytes = dim * sizeof(float);

    float* arrOut, * d_arrIn, * d_arrOut;
    CHECK(hipMalloc((void**)&d_arrIn, arrayBytes));
    CHECK(hipMemcpy(d_arrIn, arrIn, arrayBytes, hipMemcpyHostToDevice));
    CHECK(hipMalloc((void**)&d_arrOut, blocksBytes));
    CHECK(hipMemset((void*)d_arrOut, 0, blocksBytes));
    arrOut = (float*)malloc(blocksBytes * sizeof(float));

    SumParReduce << <numBlocks, BLOCKSIZE >> > (d_arrIn, d_arrOut, dim);
    CHECK(hipDeviceSynchronize());
    CHECK(hipGetLastError());

    // memcopy D2H
    CHECK(hipMemcpy(arrOut, d_arrOut, blocksBytes, hipMemcpyDeviceToHost));

    // check result
    for (uint i = 0; i < numBlocks; i++) {
        sum += arrOut[i];
    }

    hipFree(d_arrOut);
    hipFree(d_arrIn);
    free(arrOut);

    CHECK(hipDeviceReset());

    return sum;
}

//  def normpdf(x, mu=0, sigma=1):
static float normpdf(const float x, const float mu = 0.0f, const float sigma = 1.0f) {
    //  u = float((x - mu) / abs(sigma))
    //  y = exp(-u * u / 2) / (sqrt(2 * pi) * abs(sigma))
    //  return y
    float u = (x - mu) / abs(sigma);
    float y = exp(-u * u / 2) / (sqrt(PI2) * abs(sigma));
    return y;
}
__device__ float normPdfGPU(const float x, const float mu = 0.0f, const float sigma = 1.0f) {
    float u = (x - mu) / abs(sigma);
    float y = exp(-u * u / 2) / (sqrt(PI2) * abs(sigma));
    return y;
}

float sqrdMagnitude(const float* const X, const int dim) {
    float sqrdMag = FLT_EPSILON;
    for (int i = 0; i < dim; i++) {
        sqrdMag += X[i] * X[i];
    }
    return sqrdMag;
}
static float magnitudeXY(const Particles* const p) {
    float sqrdMagX = sqrdMagnitude(p->x, p->size);
    float sqrdMagY = sqrdMagnitude(p->y, p->size);
    float magnitude = sqrt(sqrdMagX + sqrdMagY);
    return magnitude;
}
__device__ float MagnitudeGPU(const float x, const float y) {
    float mag = sqrt((x * x) + (y * y));
    return mag;
}
__device__ float MagnitudeGPU(const Float2* const vec2) {
    return MagnitudeGPU(vec2->x, vec2->y);

}
__device__ float MagnitudeGPU(const Float2 vec2) {
    return MagnitudeGPU(vec2.x, vec2.y);
}

static Float2 WeightedAverage(const Floats2* const pos, const float* const weights, const int dim) {
    Float2 avg;
    avg.x = 0.0f;
    avg.y = 0.0f;

    // numpy implementation: avg = sum(a * weights) / sum(weights)

    float sumWeights = FLT_EPSILON; // avoid div by 0
    Float2 sumPos;
    sumPos.x = 0.0f;
    sumPos.y = 0.0f;
    for (int i = 0; i < dim; i++) {
        sumWeights += weights[i];
        sumPos.x += pos->x[i] * weights[i];
        sumPos.y += pos->y[i] * weights[i];
    }

    avg.x = sumPos.x / sumWeights;
    avg.y = sumPos.y / sumWeights;

    return avg;
}

static float* CumSum(const float* const arr_in, const int dim) {
    float* cumSumArr = (float*)malloc(dim * sizeof(float));

    for (int i = 0; i < dim; i++) {
        for (int j = 0; j <= i; j++) {
            cumSumArr[i] += arr_in[j];
        }
    }

    return cumSumArr;
}

static float* CumSumGPU(const float* const arr_in, const int dim) {
    float* cumSumArr = (float*)malloc(dim * sizeof(float));

    for (int i = 0; i < dim; i++) {
        cumSumArr[i] = SumArrayGPU(arr_in, i + 1);  // + 1 for the size of the subArr
    }

    return cumSumArr;
}

// find the index, in the sorted array (ascending order), to insert the element preserving the order
static int SearchSorted(const float* const sortedArry, const float element, const int dim) {
    int index = dim;

    for (int i = 0; i < dim - 1; i++) {
        if (element <= sortedArry[i] && element > sortedArry[i + 1]) {
            index = i;
        }
    }

    return index;
}

__global__ void DivisionKernel(float* const dividend, const uint dividendDim, const float divisor) {
    uint tid = threadIdx.x;
    ulong idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= dividendDim)
        return;

    dividend[idx] /= divisor;
}
static void ParallelDivisionGPU(float* const dividend, const uint dividendDim, const float divisor) {
    uint numBlocks = (dividendDim + BLOCKSIZE - 1) / BLOCKSIZE;

    long arrayBytes = dividendDim * sizeof(float);

    float* d_dividend;
    CHECK(hipMalloc((void**)&d_dividend, arrayBytes));
    CHECK(hipMemcpy(d_dividend, dividend, arrayBytes, hipMemcpyHostToDevice));

    DivisionKernel << <numBlocks, BLOCKSIZE >> > (d_dividend, dividendDim, divisor);

    CHECK(hipDeviceSynchronize());
    CHECK(hipGetLastError());

    // memcopy D2H
    CHECK(hipMemcpy(dividend, d_dividend, arrayBytes, hipMemcpyDeviceToHost));

    hipFree(d_dividend);

    CHECK(hipDeviceReset());
}


static void PredictCPU(Particles* const p, const Float2* const u, const Float2* const std, const float dt) {
    //""" move according to control input u (heading change, velocity)
    //    with noise Q(std heading change, std velocity)`"""
    srand((unsigned int)time(NULL));   // Initialization, should only be called once.

    for (int i = 0; i < p->size; i++) {
        float r = ((float)rand() / (float)(RAND_MAX));      // rand Returns a pseudo-random integer between 0 and RAND_MAX.

        // update heading
        p->heading[i] += u->x + (r * std->x);
        p->heading[i] = fmodf(p->heading[i], PI2);

        float dist = (u->y * dt) + (r * std->y);

        // move in the(noisy) commanded direction
        p->x[i] += cos(p->heading[i]) * dist;
        p->y[i] += sin(p->heading[i]) * dist;
    }
}

static void PredictGPU(Particles* const p, const Float2* const u, const Float2* const std, const float dt) {
    long particlesBytes = BytesOfParticles(p);

    Particles* d_particlesIn;
    CHECK(hipMalloc((void**)&d_particlesIn, particlesBytes));
    CHECK(hipMemcpy(d_particlesIn, p, particlesBytes, hipMemcpyHostToDevice));

    Particles* d_particlesOut;
    CHECK(hipMalloc((void**)&d_particlesOut, particlesBytes));
    CHECK(hipMemcpy(d_particlesOut, p, particlesBytes, hipMemcpyHostToDevice));

    uint numBlocks = (p->size + BLOCKSIZE - 1) / BLOCKSIZE;
    hiprandState* devStates;

    //__global__ void PredictGPUKernel(Particles * D_in, Particles * C_out, hiprandState * states, float* u, float* std, float dt) {
    PredictGPUKernel << <numBlocks, BLOCKSIZE >> > (d_particlesIn, d_particlesOut, devStates, (*u), (*std), dt);

    CHECK(hipDeviceSynchronize());

    CHECK(hipMemcpy(p, d_particlesOut, particlesBytes, hipMemcpyDeviceToHost));
    CHECK(hipGetLastError());

    CHECK(hipFree(d_particlesIn));
    CHECK(hipFree(d_particlesOut));
}

//def update(particles, weights, z, R, landmarks) :
//  for i, landmark in enumerate(landmarks) :
//      distance = np.linalg.norm(particles[:, 0 : 2] - landmark, axis = 1)
//      weights *= scipy.stats.norm(distance, R).pdf(z[i])
//
//  weights += 1.e-300      # avoid round - off to zero
//  weights /= sum(weights) # normalize
static void UpdateCPU(Particles* const p, const float const* z, const float R, const Floats2 const* landmarks, const int numberOfLandmarks) {
    int size = p->size;

    Floats2 distance;
    distance.x = (float*)malloc(size * sizeof(float));
    distance.y = (float*)malloc(size * sizeof(float));
    float* normPdfs = (float*)malloc(size * sizeof(float));
    float* distanceMagnitudes = (float*)calloc(size, sizeof(float));

    for (int i = 0; i < numberOfLandmarks; i++) {
        //  distance = np.linalg.norm(particles[:, 0 : 2] - landmark, axis = 1)
        memcpy(distance.x, p->x, size * sizeof(float));
        memcpy(distance.y, p->y, size * sizeof(float));

        for (int j = 0; j < size; j++) {    // particles[:, 0 : 2] - landmark
            distance.x[j] -= landmarks->x[i];
            distance.y[j] -= landmarks->y[i];
        }

        for (int j = 0; j < size; j++) {    // np.linalg.norm
            distanceMagnitudes[j] = Magnitude(distance.x[j], distance.y[j]);
        }

        //  weights *= scipy.stats.norm(distance, R).pdf(z[i])
        for (int j = 0; j < size; j++) { // scipy.stats.norm(distance, R).pdf(z[i])
            normPdfs[j] = normpdf(z[i], distanceMagnitudes[j], R);;
        }
        for (int j = 0; j < size; j++) { // weights *=  // element wise multiplication
            p->weights[j] *= normPdfs[j];
        }
    }

    free(distanceMagnitudes);
    free(normPdfs);
    free(distance.x);
    free(distance.y);

    float sum = FLT_EPSILON;  // avoid round - off to zero
    for (int i = 0; i < size; i++) {
        sum += p->weights[i];
    }

    for (int i = 0; i < size; i++) {
        p->weights[i] /= sum; // normalize
    }
}

//# def update(particles, weights, z, R, landmarks):
__global__ void UpdateGPUKernel(Particles* const p, float* distanceX, float* distanceY, const float const* z, const float R, const Float2 landmark) {
    uint tid = threadIdx.x;
    ulong idx = blockIdx.x * blockDim.x + threadIdx.x;

    // boundary check
    if (idx >= p->size)
        return;

    //for (int j = 0; j < size; j++) {    // particles[:, 0 : 2] - landmark
    //    distance.x[j] -= landmarks->x[i];
    //    distance.y[j] -= landmarks->y[i];
    //}
    distanceX[idx] -= landmark.x;
    distanceY[idx] -= landmark.y;
  
    //for (int j = 0; j < size; j++) {    // np.linalg.norm
    //    distanceMagnitudes[j] = Magnitude(distanceX[j], distanceY[j]);
    //}
    float distanceMagnitude = MagnitudeGPU(distanceX[idx], distanceY[idx]);

    //  weights *= scipy.stats.norm(distance, R).pdf(z[i])
    //for (int j = 0; j < size; j++) { // scipy.stats.norm(distance, R).pdf(z[i])
    //    normPdfs[j] = normpdf(z[i], distanceMagnitudes[j], R);;
    //}
    float normPdf = normPdfGPU(z[idx], distanceMagnitude, R);

    //for (int j = 0; j < size; j++) { // weights *=  // element wise multiplication
    //    p->weights[j] *= normPdfs[j];
    //}
    p->weights[idx] *= normPdf;
}

static void UpdateGPU(Particles* const p, const float const* z, const float R, const Floats2 const* landmarks, const int numberOfLandmarks) {
    int size = p->size;

    long particlesBytes = BytesOfParticles(p);

    Particles* d_particles;
    CHECK(hipMalloc((void**)&d_particles, particlesBytes));
    CHECK(hipMemcpy(d_particles, p, particlesBytes, hipMemcpyHostToDevice));

    ulong arrayBytes = p->size * sizeof(float);
    long landmarkBytes = numberOfLandmarks * sizeof(float);

    float* d_distanceX, * d_distanceY;
    CHECK(hipMalloc((void**)&d_distanceX, arrayBytes));
    CHECK(hipMalloc((void**)&d_distanceY, arrayBytes));

    float* d_z;
    CHECK(hipMalloc((void**)&d_z, arrayBytes));
    CHECK(hipMemcpy(d_z, z, landmarkBytes, hipMemcpyHostToDevice));

    uint numBlocks = (p->size + BLOCKSIZE - 1) / BLOCKSIZE;

    for (int i = 0; i < numberOfLandmarks; i++) {
        Float2 landmark;
        landmark.x = landmarks->x[i];
        landmark.y = landmarks->y[i];
        
        CHECK(hipMemcpy(d_distanceX, p->x, arrayBytes, hipMemcpyHostToDevice));
        CHECK(hipMemcpy(d_distanceY, p->y, arrayBytes, hipMemcpyHostToDevice));

        UpdateGPUKernel << <numBlocks, BLOCKSIZE >> > (d_particles, d_distanceX, d_distanceY, d_z, R, landmark);

        CHECK(hipDeviceSynchronize());

        // Update weights
        CHECK(hipMemcpy(p, d_particles, particlesBytes, hipMemcpyDeviceToHost));
        CHECK(hipGetLastError());
    }

    CHECK(hipFree(d_particles));
    CHECK(hipFree(d_distanceX));
    CHECK(hipFree(d_distanceY));
    CHECK(hipFree(d_z));

    // Normalization
    float sum = FLT_EPSILON;  // avoid round - off to zero
    sum = SumArrayGPU(p->weights, p->size);
    ParallelDivisionGPU(p->weights, p->size, sum);

    CHECK(hipDeviceReset());
}

//def estimate(particles, weights) :
//    """returns mean and variance of the weighted particles"""
//
//    pos = particles[:, 0 : 2]
//    mean = np.average(pos, weights = weights, axis = 0)
//    var = np.average((pos - mean) * *2, weights = weights, axis = 0)
//    return mean, var
// returns mean and variance of the weighted particles
static void EstimateCPU(const Particles* const p, Float2* const mean_out, Float2* const var_out) {

    Floats2 pos;
    pos.x = (float*)malloc(p->size * sizeof(float));
    memcpy(pos.x, p->x, p->size * sizeof(float));
    pos.y = (float*)malloc(p->size * sizeof(float));
    memcpy(pos.y, p->y, p->size * sizeof(float));

    // mean = np.average(pos, weights = weights, axis = 0)
    (*mean_out) = WeightedAverage(&pos, p->weights, p->size);
    // var = np.average((pos - mean) **2, weights = weights, axis = 0)
    for (int i = 0; i < p->size; i++) {
        pos.x[i] = (pos.x[i] - mean_out->x) * (pos.x[i] - mean_out->x);
        pos.y[i] = (pos.y[i] - mean_out->y) * (pos.y[i] - mean_out->y);
    }

    (*var_out) = WeightedAverage(&pos, p->weights, p->size);

    free(pos.x);
    free(pos.y);
}
static void EstimateGPU(const Particles* const p, Float2* const mean_out, Float2* const var_out) {

    Floats2 pos;
    pos.x = (float*)malloc(p->size * sizeof(float));
    memcpy(pos.x, p->x, p->size * sizeof(float));
    pos.y = (float*)malloc(p->size * sizeof(float));
    memcpy(pos.y, p->y, p->size * sizeof(float));

    // mean = np.average(pos, weights = weights, axis = 0)
    (*mean_out) = WeightedAverage(&pos, p->weights, p->size);
    // var = np.average((pos - mean) **2, weights = weights, axis = 0)
    for (int i = 0; i < p->size; i++) {
        pos.x[i] = (pos.x[i] - mean_out->x) * (pos.x[i] - mean_out->x);
        pos.y[i] = (pos.y[i] - mean_out->y) * (pos.y[i] - mean_out->y);
    }

    (*var_out) = WeightedAverage(&pos, p->weights, p->size);

    free(pos.x);
    free(pos.y);
}

//def simple_resample(particles, weights) :
//    N = len(particles)
//    cumulative_sum = np.cumsum(weights)
//    cumulative_sum[-1] = 1. # avoid round - off error
//    indexes = np.searchsorted(cumulative_sum, random(N))  // Return random floats in the half-open interval [0.0, 1.0).
//
//    # resample according to indexes
//    particles[:] = particles[indexes]
//    weights.fill(1.0 / N)
static void SimpleResample(Particles* const p) {
    int dim = p->size;

    //    cumulative_sum = np.cumsum(weights)
    float* cumSum_arr = CumSum(p->weights, dim);

    // indexes = np.searchsorted(cumulative_sum, random(N))  // Return random floats in the half-open interval [0.0, 1.0).
    int* indexes = (int*)malloc(dim * sizeof(int));
    srand((unsigned int)time(NULL));   // Initialization, should only be called once.
    float r = 0.0f;
    for (int i = 0; i < dim; i++) {
        r = ((float)rand() / (float)(RAND_MAX));      // rand Returns a pseudo-random integer between 0 and RAND_MAX.
        indexes[i] = SearchSorted(cumSum_arr, r, dim);
    }

    //  # resample according to indexes
    //  particles[:] = particles[indexes]
    //  weights.fill(1.0 / N)
    float equalWeight = 1.0f / dim;
    for (int i = 0; i < dim; i++) {
        p->x[i] = p->x[indexes[i]];
        p->y[i] = p->y[indexes[i]];
        p->heading[i] = p->heading[indexes[i]];
        p->weights[i] = equalWeight;
    }

    free(indexes);
    free(cumSum_arr);
}

// We don't resample at every epoch. 
// For example, if you received no new measurements you have not received any information from which the resample can benefit. 
// We can determine when to resample by using something called the *effective N*, 
// which approximately measures the number of particles which meaningfully contribute to the probability distribution.
//def neff(weights) :
//    return 1. / np.sum(np.square(weights))
static float Neff(const float* const weights, const int dim) {
    float res = 0.0f;
    float sum = FLT_EPSILON;

    for (int i = 0; i < dim; i++) {
        float squaredWeight = weights[i] * weights[i];
        sum += squaredWeight;
    }

    res = 1.0f / sum;
    return res;
}

static float NeffGPU(const float* const weightsIn, const int dim) {
    float res = 0.0f;
    float sum = FLT_EPSILON;

    int blockSize = 1024;            // block dim 1D
    //int numBlock = 1024 * 1024;      // grid dim 1D
    int numBlock = (dim / blockSize);
    if (dim % blockSize != 0) {
        numBlock += 1;
    }

    long blocksBytes = numBlock * sizeof(float);
    long arrayBytes = dim * sizeof(float);

    float* weightsOut, * d_weightsIn, * d_weightsOut;
    CHECK(hipMalloc((void**)&d_weightsIn, arrayBytes));
    CHECK(hipMemcpy(d_weightsIn, weightsIn, arrayBytes, hipMemcpyHostToDevice));
    CHECK(hipMalloc((void**)&d_weightsOut, blocksBytes));
    CHECK(hipMemset((void*)d_weightsOut, 0, blocksBytes));
    weightsOut = (float*)malloc(blocksBytes * sizeof(float));

    SumSquaredParReduce << <numBlock, blockSize >> > (d_weightsIn, d_weightsOut, dim);
    CHECK(hipDeviceSynchronize());
    CHECK(hipGetLastError());

    // memcopy D2H
    CHECK(hipMemcpy(weightsOut, d_weightsOut, blocksBytes, hipMemcpyDeviceToHost));

    // check result
    for (uint i = 0; i < numBlock; i++) {
        sum += weightsOut[i];
    }

    res = 1.0f / sum;

    hipFree(d_weightsOut);
    hipFree(d_weightsIn);
    free(weightsOut);

    CHECK(hipDeviceReset());

    return res;
}

/*
 *  Device function: block parallel reduction based on warp unrolling
 */
__device__ void blockWarpUnroll(float* thisBlock, int blockDim, uint tid) {
    // in-place reduction in global memory
    for (int stride = blockDim / 2; stride > 32; stride >>= 1) {
        if (tid < stride)
            thisBlock[tid] += thisBlock[tid + stride];

        // synchronize within threadblock
        __syncthreads();
    }

    // unrolling warp
    if (tid < 32) {
        volatile float* vmem = thisBlock;
        vmem[tid] += vmem[tid + 32];
        vmem[tid] += vmem[tid + 16];
        vmem[tid] += vmem[tid + 8];
        vmem[tid] += vmem[tid + 4];
        vmem[tid] += vmem[tid + 2];
        vmem[tid] += vmem[tid + 1];
    }
}

__global__ void Norm_BlockUnroll8(float* in, float* out, const float add, const ulong n) {
    uint tid = threadIdx.x;
    ulong idx = blockIdx.x * blockDim.x + threadIdx.x;

    // boundary check
    if (idx >= n)
        return;

    // convert global data pointer to the local pointer of this block
    float* thisBlock = in + blockIdx.x * blockDim.x * 8;

    float a = 0.0f;
    float temp = 0.0f;

    // unrolling 8 blocks
    if (idx + 7 * blockDim.x < n) {
        for (int i = 0; i < 8; i++) {
            temp = in[idx + i * blockDim.x];
            temp += add;
            a += temp * temp;
        }
        in[idx] = a;
    }

    __syncthreads();

    // block parall. reduction based on warp unrolling 
    blockWarpUnroll(thisBlock, blockDim.x, tid);

    // write result for this block to global mem
    if (tid == 0)
        out[blockIdx.x] = thisBlock[0];
}


void particleFilterGPU(Particles* const p, const int iterations, const float sensorStdError) {

    Particles d_p;

    CHECK(hipMalloc((void**)&d_p.x, DIM * sizeof(float)));
    CHECK(hipMemcpy(d_p.x, p->x, DIM * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMalloc((void**)&d_p.y, DIM * sizeof(float)));
    CHECK(hipMemcpy(d_p.y, p->y, DIM * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMalloc((void**)&d_p.weights, DIM * sizeof(float)));
    CHECK(hipMemcpy(d_p.weights, p->weights, DIM * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMalloc((void**)&d_p.heading, DIM * sizeof(float)));
    CHECK(hipMemcpy(d_p.heading, p->heading, DIM * sizeof(float), hipMemcpyHostToDevice));

    // OUTPUT for Device
    Particles d_out;
    CHECK(hipMalloc((void**)&d_out.x, DIM * sizeof(float)));
    CHECK(hipMemcpy(d_out.x, p->x, DIM * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMalloc((void**)&d_out.y, DIM * sizeof(float)));
    CHECK(hipMemcpy(d_out.y, p->y, DIM * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMalloc((void**)&d_out.weights, DIM * sizeof(float)));
    CHECK(hipMemcpy(d_out.weights, p->weights, DIM * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMalloc((void**)&d_out.heading, DIM * sizeof(float)));
    CHECK(hipMemcpy(d_out.heading, p->heading, DIM * sizeof(float), hipMemcpyHostToDevice));


    hipFree(d_p.x);
    hipFree(d_p.y);
    hipFree(d_p.heading);
    hipFree(d_p.weights);
    hipFree(d_out.x);
    hipFree(d_out.y);
    hipFree(d_out.heading);
    hipFree(d_out.weights);
}

void particleFilterCPU(Particles* const p, const int iterations, const float sensorStdError) {
    clock_t start, stop;
    double timer;

    printf(" - particleFilterC - \n");

    unsigned int dim = p->size;

    printf("number of particles: %d \n", dim);

    start = clock();

    // Start

    Float2 u;
    Float2 std;
    float dt = 0.1f;

    Floats2 landmarks;
    int numberOfLandmarks = 4;
    landmarks.x = (float*)malloc(numberOfLandmarks * sizeof(float));
    landmarks.y = (float*)malloc(numberOfLandmarks * sizeof(float));

    landmarks.x[0] = -1.0f;
    landmarks.y[0] = 2.0f;

    landmarks.x[1] = 5.0f;
    landmarks.y[1] = 10.0f;

    landmarks.x[2] = 12.0f;
    landmarks.y[2] = 24.0f;

    landmarks.x[3] = 18.0f;
    landmarks.y[3] = 21.0f;

    Float2 robotPosition;
    robotPosition.x = 0.0f;
    robotPosition.y = 0.0f;

    Float2* xs;
    xs = (Float2*)malloc(iterations * sizeof(Float2));

    for (int i = 0; i < iterations; i++) {
        // Diagonal movement
        robotPosition.x += 1.0f;
        robotPosition.y += 1.0f;

        srand((unsigned int)time(NULL));   // Initialization, should only be called once.
        float r = 0.0f;
        float* zs = (float*)malloc(numberOfLandmarks * sizeof(float));
        for (int j = 0; j < numberOfLandmarks; j++) {
            Float2 landmark;
            landmark.x = landmarks.x[j];
            landmark.y = landmarks.y[j];
            Float2 distanceRobotLandmark = Minus(&landmark, &robotPosition);
            float magnitudeDistance = Magnitude(distanceRobotLandmark);
            r = ((float)rand() / (float)(RAND_MAX));      // rand Returns a pseudo-random integer between 0 and RAND_MAX.
            zs[j] = magnitudeDistance + (r * sensorStdError);
        }

        PredictCPU(p, &u, &std, dt);

        UpdateCPU(p, zs, sensorStdError, &landmarks, numberOfLandmarks);

        //# resample if too few effective particles
        //    if neff(weights) < N / 2:
        //indexes = systematic_resample(weights)
        //    resample_from_index(particles, weights, indexes)
        //    assert np.allclose(weights, 1 / N)
        //mu, var = estimate(particles, weights)
        //xs.append(mu)

        float neff = Neff(p->weights, p->size);
        if (neff < p->size / 2.0f) {
            // resample
            SimpleResample(p);
        }

        Float2 var;
        Float2 mean;

        EstimateCPU(p, &mean, &var);

        xs[i] = mean;

        free(zs);
    }

    // End

    stop = clock();
    timer = ((double)(stop - start)) / (double)CLOCKS_PER_SEC;
    printf("\n\n Total execution time: %9.4f sec \n\n", timer);

    free(landmarks.x);
    free(landmarks.y);
    free(xs);
}

void euclideanNorm(Particles* p, float2* norm, float2* landmark) {

    Particles d_p;

    CHECK(hipMalloc((void**)&d_p.x, DIM * sizeof(float)));
    CHECK(hipMemcpy(d_p.x, p->x, DIM * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMalloc((void**)&d_p.y, DIM * sizeof(float)));
    CHECK(hipMemcpy(d_p.y, p->y, DIM * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMalloc((void**)&d_p.weights, DIM * sizeof(float)));
    CHECK(hipMemcpy(d_p.weights, p->weights, DIM * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMalloc((void**)&d_p.heading, DIM * sizeof(float)));
    CHECK(hipMemcpy(d_p.heading, p->heading, DIM * sizeof(float), hipMemcpyHostToDevice));


    // OUTPUT for Device
    Particles d_out;
    CHECK(hipMalloc((void**)&d_out.x, DIM * sizeof(float)));
    CHECK(hipMemcpy(d_out.x, p->x, DIM * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMalloc((void**)&d_out.y, DIM * sizeof(float)));
    CHECK(hipMemcpy(d_out.y, p->y, DIM * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMalloc((void**)&d_out.weights, DIM * sizeof(float)));
    CHECK(hipMemcpy(d_out.weights, p->weights, DIM * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMalloc((void**)&d_out.heading, DIM * sizeof(float)));
    CHECK(hipMemcpy(d_out.heading, p->heading, DIM * sizeof(float), hipMemcpyHostToDevice));


    Particles p_norm;

    Create_Particles(&p_norm, DIM);

    Norm_BlockUnroll8 << <NUMBLOCKS / 8, BLOCKSIZE >> > (d_p.x, d_out.x, -landmark->x, N);  // ERRROR at <<< can be simply ignored
    Norm_BlockUnroll8 << <NUMBLOCKS / 8, BLOCKSIZE >> > (d_p.y, d_out.y, -landmark->y, N);

    CHECK(hipDeviceSynchronize());
    CHECK(hipGetLastError());

    CHECK(hipMemcpy(p_norm.x, d_out.x, DIM * sizeof(float), hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(p_norm.y, d_out.y, DIM * sizeof(float), hipMemcpyDeviceToHost));

    // Sum on CPU the last elements
    for (uint i = 0; i < NUMBLOCKS / 8; i++) {
        norm->x += p_norm.x[i];
        norm->y += p_norm.y[i];
    }

    hipFree(d_p.x);
    hipFree(d_p.y);
    hipFree(d_p.heading);
    hipFree(d_p.weights);
    hipFree(d_out.x);
    hipFree(d_out.y);
    hipFree(d_out.heading);
    hipFree(d_out.weights);
    free(p_norm.x);
    free(p_norm.y);
    free(p_norm.heading);
    free(p_norm.weights);
}

int main()
{
    Float2 xRange;
    xRange.x = MinX;
    xRange.y = MaxX;

    Float2 yRange;
    yRange.x = MinY;
    yRange.y = MaxY;

    Float2 headingRange;
    headingRange.x = MinHeading;
    headingRange.y = MaxHeading;

    Particles p;

    CreateAndRandomInitialize_Particles(&p, DIM, &xRange, &yRange, &headingRange);

    //particleFilterGPU(&p);

    particleFilterCPU(&p, 18, 0.1f);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    CHECK(hipDeviceReset());

    return 0;
}


//// Helper function for using CUDA to add vectors in parallel.
//hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
//{
//    int *dev_a = 0;
//    int *dev_b = 0;
//    int *dev_c = 0;
//    hipError_t cudaStatus;
//
//    //// Choose which GPU to run on, change this on a multi-GPU system.
//    cudaStatus = hipSetDevice(0);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
//        goto Error;
//    }
//
//    // Allocate GPU buffers for three vectors (two input, one output)    .
//    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMalloc failed!");
//        goto Error;
//    }
//
//    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMalloc failed!");
//        goto Error;
//    }
//
//    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMalloc failed!");
//        goto Error;
//    }
//
//    // Copy input vectors from host memory to GPU buffers.
//    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMemcpy failed!");
//        goto Error;
//    }
//
//    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMemcpy failed!");
//        goto Error;
//    }
//
//    // Launch a kernel on the GPU with one thread for each element.
//    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);
//
//    // Check for any errors launching the kernel
//    cudaStatus = hipGetLastError();
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
//        goto Error;
//    }
//    
//    // hipDeviceSynchronize waits for the kernel to finish, and returns
//    // any errors encountered during the launch.
//    cudaStatus = hipDeviceSynchronize();
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
//        goto Error;
//    }
//
//    // Copy output vector from GPU buffer to host memory.
//    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMemcpy failed!");
//        goto Error;
//    }
//
//Error:
//    hipFree(dev_c);
//    hipFree(dev_a);
//    hipFree(dev_b);
//    
//    return cudaStatus;
//}

