#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#include "common.h"
#include "Particle.h"

#include <stdio.h>
#include <cmath>

#define PI 3.141592f

#define IDX(i,j,n) (i*n+j)
#define ABS(x,y) (x-y>=0?x-y:y-x)
#define DIM 100


hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);



__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

__global__ void GenerateParticles(Particle* D_in, Particle* C_out, hiprandState* states, Vec2 x_range, Vec2 y_range, Vec2 heading_range) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    hiprand_init(tid, 0, 0, &states[tid]);
    float pos_x = x_range.x + ((x_range.y - x_range.x) * hiprand_uniform(&states[tid]));
    float pos_y = y_range.x + ((y_range.y - y_range.x) * hiprand_uniform(&states[tid]));
    float heading = hiprand_uniform(&states[tid]);
    heading = std::fmod(heading, 2 * PI);

    C_out->x[tid] = pos_x;
    C_out->y[tid] = pos_y;
    C_out->heading[tid] = heading;
}

//# def predict(particles, u, std, dt=1.):
//# """ move according to control input u (heading change, velocity)
//# with noise Q (std heading change, std velocity)`"""
__global__ void Predict(Particle* D_in, Particle* C_out, hiprandState* states, float* u, float* std, float dt) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    dt = 1.0f;

    hiprand_init(tid, 0, 0, &states[tid]);

    //# update heading
    float heading = D_in->heading[tid];
    heading += u[0] + hiprand_normal(&states[tid]);
    heading = std::fmod(D_in->heading[tid], 2 * PI);

    //# move in the (noisy) commanded direction
    float dist = (u[1] * dt) + (hiprand_uniform(&states[tid]) * std[1]);
    float pos_x = D_in->x[tid];
    float pos_y = D_in->y[tid];
    pos_x += std::cos(D_in->heading[tid]) * dist;
    pos_y += std::sin(D_in->heading[tid]) * dist;

    C_out->x[tid] = pos_x;
    C_out->y[tid] = pos_y;
    C_out->heading[tid] = heading;
}

//# def update(particles, weights, z, R, landmarks):
__global__ void Update(Particle* particles, Particle* C_out, float* weights, float* z, float* landmarks, int numberOfLandmarks, float R) {
    //# weights init as ones / N

    for (int i = 0; i < numberOfLandmarks; i++) {
        float distance =
    }

    for i, landmark in enumerate(landmarks) :
        distance = np.linalg.norm(particles[:, 0 : 2] - landmark, axis = 1)
        weights *= scipy.stats.norm(distance, R).pdf(z[i])

        weights += 1.e-300      # avoid round - off to zero
        weights /= sum(weights) # normalize
}

int main()
{
    Particle p;

    CreateParticleDim(&p, DIM);

    hipError_t cudaStatus = particleFilter(&p);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Particle filter failed!");
        return 1;
    }

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

hipError_t particleFilter(Particle* p) {
    hipError_t cudaStatus;
    
    //// Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    Particle d_p;

    //CHECK(hipMalloc((void**)&d_p.x, DIM * sizeof(float)));
    //CHECK(hipMemcpy(d_p.x, p->x, DIM * sizeof(float), hipMemcpyHostToDevice));
    //CHECK(hipMalloc((void**)&d_p.y, DIM * sizeof(float)));
    //CHECK(hipMemcpy(d_p.y, p.y, DIM * sizeof(float), hipMemcpyHostToDevice));
    //CHECK(hipMalloc((void**)&d_p.weight, DIM * sizeof(float)));
    //CHECK(hipMemcpy(d_p.weight, p.weight, DIM * sizeof(float), hipMemcpyHostToDevice));
    //CHECK(hipMalloc((void**)&d_p.heading, DIM * sizeof(float)));
    //CHECK(hipMemcpy(d_p.heading, p.heading, DIM * sizeof(float), hipMemcpyHostToDevice));

    cudaStatus = hipMalloc((void**)&d_p.x, DIM * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMemcpy(d_p.x, p->x, DIM * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&d_p.y, DIM * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMemcpy(d_p.y, p->y, DIM * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&d_p.weight, DIM * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMemcpy(d_p.weight, p->weight, DIM * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&d_p.heading, DIM * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMemcpy(d_p.heading, p->heading, DIM * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    //// Launch a kernel on the GPU with one thread for each element.
    //addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    //// Check for any errors launching the kernel
    //cudaStatus = hipGetLastError();
    //if (cudaStatus != hipSuccess) {
    //    fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
    //    goto Error;
    //}
    //
    //// hipDeviceSynchronize waits for the kernel to finish, and returns
    //// any errors encountered during the launch.
    //cudaStatus = hipDeviceSynchronize();
    //if (cudaStatus != hipSuccess) {
    //    fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
    //    goto Error;
    //}

    //// Copy output vector from GPU buffer to host memory.
    //cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    //if (cudaStatus != hipSuccess) {
    //    fprintf(stderr, "hipMemcpy failed!");
    //    goto Error;
    //}


Error:
    hipFree(d_p.x);
    hipFree(d_p.y);
    hipFree(d_p.weight);
    hipFree(d_p.heading);

    return cudaStatus;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    //// Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    //// Allocate GPU buffers for three vectors (two input, one output)    .
    //cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    //if (cudaStatus != hipSuccess) {
    //    fprintf(stderr, "hipMalloc failed!");
    //    goto Error;
    //}

    //cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    //if (cudaStatus != hipSuccess) {
    //    fprintf(stderr, "hipMalloc failed!");
    //    goto Error;
    //}

    //cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    //if (cudaStatus != hipSuccess) {
    //    fprintf(stderr, "hipMalloc failed!");
    //    goto Error;
    //}

    //// Copy input vectors from host memory to GPU buffers.
    //cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    //if (cudaStatus != hipSuccess) {
    //    fprintf(stderr, "hipMemcpy failed!");
    //    goto Error;
    //}

    //cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    //if (cudaStatus != hipSuccess) {
    //    fprintf(stderr, "hipMemcpy failed!");
    //    goto Error;
    //}

    //// Launch a kernel on the GPU with one thread for each element.
    //addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    //// Check for any errors launching the kernel
    //cudaStatus = hipGetLastError();
    //if (cudaStatus != hipSuccess) {
    //    fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
    //    goto Error;
    //}
    //
    //// hipDeviceSynchronize waits for the kernel to finish, and returns
    //// any errors encountered during the launch.
    //cudaStatus = hipDeviceSynchronize();
    //if (cudaStatus != hipSuccess) {
    //    fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
    //    goto Error;
    //}

    //// Copy output vector from GPU buffer to host memory.
    //cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    //if (cudaStatus != hipSuccess) {
    //    fprintf(stderr, "hipMemcpy failed!");
    //    goto Error;
    //}

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
